#include "hip/hip_runtime.h"
/**
 * @file poisson_adaptive.cu
 * @author Cu Cui (cu.cui@iwr.uni-heidelberg.de)
 * @brief Discontinuous Galerkin methods for poisson problems with local refinement.
 * @version 1.0
 * @date 2023-02-02
 *
 * @copyright Copyright (c) 2023
 *
 */

#include <deal.II/base/conditional_ostream.h>
#include <deal.II/base/convergence_table.h>
#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/function.h>
#include <deal.II/base/function_lib.h>
#include <deal.II/base/quadrature_lib.h>
#include <deal.II/base/timer.h>

#include <deal.II/distributed/grid_refinement.h>
#include <deal.II/distributed/tria.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_dgq.h>
#include <deal.II/fe/fe_q.h>
#include <deal.II/fe/mapping_q.h>

#include <deal.II/grid/filtered_iterator.h>
#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/grid_refinement.h>
#include <deal.II/grid/grid_tools.h>
#include <deal.II/grid/manifold_lib.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/la_parallel_vector.h>

#include <deal.II/numerics/data_out.h>
#include <deal.II/numerics/vector_tools.h>

#include <hip/hip_runtime_api.h>

#include <fstream>

#include "app_utilities.h"
#include "ct_parameter.h"
#include "cuda_fe_evaluation.cuh"
#include "cuda_matrix_free.cuh"
#include "solver.cuh"
#include "utilities.cuh"

// -\delta u = f, u = 0 on \parital \Omege, f = 1.
// double percision

namespace Step64
{
  using namespace dealii;

  template <int dim, typename Number>
  class Solution : public Function<dim, Number>
  {
  public:
    virtual Number
    value(const Point<dim> &p, const unsigned int = 0) const override final
    {
      Number val = 1.;
      for (unsigned int d = 0; d < dim; ++d)
        val *= std::sin(numbers::PI * p[d]);
      return -val;
    }

    virtual Tensor<1, dim, Number>
    gradient(const Point<dim> &p, const unsigned int = 0) const override final
    {
      Tensor<1, dim, Number> grad;
      for (unsigned int d = 0; d < dim; ++d)
        {
          grad[d] = 1.;
          for (unsigned int e = 0; e < dim; ++e)
            if (d == e)
              grad[d] *= -numbers::PI * std::cos(numbers::PI * p[e]);
            else
              grad[d] *= std::sin(numbers::PI * p[e]);
        }
      return grad;
    }
  };

  template <int dim, typename Number>
  class RightHandSide : public Function<dim, Number>
  {
  public:
    virtual Number
    value(const Point<dim> &p, const unsigned int = 0) const override final
    {
      const Number arg = numbers::PI;
      Number       val = 1.;
      for (unsigned int d = 0; d < dim; ++d)
        val *= std::sin(arg * p[d]);
      return -dim * arg * arg * val;
    }
  };

  template <int dim, int fe_degree>
  class LaplaceProblem
  {
  public:
    using full_number   = double;
    using vcycle_number = CT::VCYCLE_NUMBER_;
    using MatrixFree    = PSMF::MatrixFree<dim, full_number>;
    using MatrixFreeDP  = PSMF::LevelVertexPatch<dim, fe_degree, full_number>;
    using MatrixFreeSP  = PSMF::LevelVertexPatch<dim, fe_degree, vcycle_number>;

    LaplaceProblem();
    ~LaplaceProblem();
    void
    run(const unsigned int n_cycles);

  private:
    void
    setup_system();
    void
    assemble_mg();
    void
    solve_mg(unsigned int n_mg_cycles);
    std::pair<double, double>
    compute_error();

    template <PSMF::LaplaceVariant  laplace,
              PSMF::LaplaceVariant  smooth_vmult,
              PSMF::SmootherVariant smooth_inv>
    void
    do_solve(unsigned int k,
             unsigned int j,
             unsigned int i,
             unsigned int call_count);

    MPI_Comm                                  mpi_communicator;
    parallel::distributed::Triangulation<dim> triangulation;
    std::shared_ptr<FiniteElement<dim>>       fe;
    DoFHandler<dim>                           dof_handler;
    MappingQ<dim>                             mapping;
    double                                    setup_time;

    std::vector<ConvergenceTable> info_table;

    std::fstream                        fout;
    std::shared_ptr<ConditionalOStream> pcout;

    std::shared_ptr<MatrixFree>                  mfdata;
    MGLevelObject<std::shared_ptr<MatrixFree>>   level_mfdata;
    MGLevelObject<std::shared_ptr<MatrixFree>>   edge_up_mfdata;
    MGLevelObject<std::shared_ptr<MatrixFree>>   edge_down_mfdata;
    MGLevelObject<std::shared_ptr<MatrixFreeDP>> patch_data_dp;
    MGLevelObject<std::shared_ptr<MatrixFreeSP>> patch_data_sp;
    MGConstrainedDoFs                            mg_constrained_dofs;
    AffineConstraints<full_number>               constraints;

    PSMF::MGTransferCUDA<dim, vcycle_number, CT::DOF_LAYOUT_> transfer;

    LinearAlgebra::distributed::Vector<full_number, MemorySpace::Host>
      ghost_solution_host;

    Vector<double> estimated_error_per_cell;
  };

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::LaplaceProblem()
    : mpi_communicator(MPI_COMM_WORLD)
    , triangulation(MPI_COMM_WORLD,
                    Triangulation<dim>::limit_level_difference_at_vertices,
                    parallel::distributed::Triangulation<
                      dim>::construct_multigrid_hierarchy)
    , fe([&]() -> std::shared_ptr<FiniteElement<dim>> {
      if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::Q)
        return std::make_shared<FE_Q<dim>>(fe_degree);
      else if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::DGQ)
        return std::make_shared<FE_DGQ<dim>>(fe_degree);
      return std::shared_ptr<FiniteElement<dim>>();
    }())
    , dof_handler(triangulation)
    , mapping(fe_degree)
    , setup_time(0.)
    , pcout(std::make_shared<ConditionalOStream>(std::cout, false))
  {
    const auto filename = Util::get_filename();
    if (Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
      {
        fout.open(filename + ".log", std::ios_base::out);
        pcout = std::make_shared<ConditionalOStream>(
          fout, Utilities::MPI::this_mpi_process(mpi_communicator) == 0);
      }

    info_table.resize(CT::LAPLACE_TYPE_.size() * CT::SMOOTH_VMULT_.size() *
                      CT::SMOOTH_INV_.size());
  }

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::~LaplaceProblem()
  {
    if (Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
      fout.close();
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::setup_system()
  {
    Timer time;
    setup_time = 0;

    dof_handler.distribute_dofs(*fe);
    dof_handler.distribute_mg_dofs();
    const unsigned int nlevels = triangulation.n_global_levels();

    auto n_replicate = CT::N_REPLICATE_;

    *pcout << "Triangulation " << triangulation.n_active_cells() << " cells, "
           << triangulation.n_levels() << " levels" << std::endl;

    *pcout << "DoFHandler " << dof_handler.n_dofs() << " dofs, level dofs";
    for (unsigned int l = 0; l < triangulation.n_levels(); ++l)
      *pcout << ' ' << dof_handler.n_dofs(l);
    *pcout << std::endl;

    constraints.clear();
    constraints.close();

    setup_time += time.wall_time();

    *pcout << "DoF setup time:         " << setup_time << "s" << std::endl;
  }
  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::assemble_mg()
  {
    // Initialization of Dirichlet boundaries
    std::set<types::boundary_id> dirichlet_boundary;
    dirichlet_boundary.insert(0);
    mg_constrained_dofs.initialize(dof_handler);
    mg_constrained_dofs.make_zero_boundary_constraints(dof_handler,
                                                       dirichlet_boundary);

    unsigned int minlevel = 0;
    unsigned int maxlevel = triangulation.n_global_levels() - 1;

    patch_data_dp.resize(minlevel, maxlevel);
    level_mfdata.resize(minlevel, maxlevel);
    edge_up_mfdata.resize(minlevel, maxlevel);
    edge_down_mfdata.resize(minlevel, maxlevel);

    if (std::is_same_v<vcycle_number, float>)
      patch_data_sp.resize(minlevel, maxlevel);

    Timer time;

    {
      typename MatrixFree::AdditionalData additional_data;
      additional_data.mapping_update_flags =
        update_values | update_gradients | update_JxW_values;
      additional_data.mapping_update_flags_inner_faces =
        update_values | update_gradients | update_JxW_values |
        update_normal_vectors;
      additional_data.matrix_type = PSMF::MatrixType::active_matrix;

      const QGauss<1> quad(fe_degree + 1);
      mfdata = std::make_shared<MatrixFree>();
      mfdata->reinit(mapping,
                     dof_handler,
                     constraints,
                     quad,
                     IteratorFilters::LocallyOwnedCell(),
                     additional_data);

      for (unsigned int level = minlevel; level <= maxlevel; ++level)
        {
          // double-precision matrix-free data
          typename MatrixFree::AdditionalData additional_data;
          additional_data.mapping_update_flags =
            update_values | update_gradients | update_JxW_values;
          additional_data.mapping_update_flags_inner_faces =
            update_values | update_gradients | update_JxW_values |
            update_normal_vectors;
          additional_data.mg_level    = level;
          additional_data.matrix_type = PSMF::MatrixType::level_matrix;

          level_mfdata[level] = std::make_shared<MatrixFree>();
          level_mfdata[level]->reinit(mapping,
                                      dof_handler,
                                      constraints,
                                      quad,
                                      IteratorFilters::LocallyOwnedLevelCell(),
                                      additional_data);

          additional_data.matrix_type = PSMF::MatrixType::edge_up_matrix;

          edge_up_mfdata[level] = std::make_shared<MatrixFree>();
          edge_up_mfdata[level]->reinit(
            mapping,
            dof_handler,
            constraints,
            quad,
            IteratorFilters::LocallyOwnedLevelCell(),
            additional_data);

          additional_data.matrix_type = PSMF::MatrixType::edge_down_matrix;

          edge_down_mfdata[level] = std::make_shared<MatrixFree>();
          edge_down_mfdata[level]->reinit(
            mapping,
            dof_handler,
            constraints,
            quad,
            IteratorFilters::LocallyOwnedLevelCell(),
            additional_data);
        }
    }

    // for (unsigned int level = minlevel; level <= maxlevel; ++level)
    //   {
    //     // double-precision matrix-free data
    //     {
    //       typename MatrixFreeDP::AdditionalData additional_data;
    //       additional_data.relaxation         = 1.;
    //       additional_data.use_coloring       = false;
    //       additional_data.patch_per_block    = CT::PATCH_PER_BLOCK_;
    //       additional_data.granularity_scheme = CT::GRANULARITY_;

    //       patch_data_dp[level] = std::make_shared<MatrixFreeDP>();
    //       patch_data_dp[level]->reinit(dof_handler, level, additional_data);
    //     }

    //     // single-precision matrix-free data
    //     if (std::is_same_v<vcycle_number, float>)
    //       {
    //         // AffineConstraints<vcycle_number> level_constraints;
    //         // level_constraints.reinit(relevant_dofs);
    //         // level_constraints.add_lines(
    //         //   mg_constrained_dofs.get_boundary_indices(level));
    //         // level_constraints.close();

    //         typename MatrixFreeSP::AdditionalData additional_data;
    //         additional_data.relaxation         = 1.;
    //         additional_data.use_coloring       = false;
    //         additional_data.patch_per_block    = CT::PATCH_PER_BLOCK_;
    //         additional_data.granularity_scheme = CT::GRANULARITY_;

    //         patch_data_sp[level] = std::make_shared<MatrixFreeSP>();
    //         patch_data_sp[level]->reinit(dof_handler, level,
    //         additional_data);
    //       }
    //   }

    *pcout << "Matrix-free setup time: " << time.wall_time() << "s"
           << std::endl;

    time.restart();

    transfer.initialize_constraints(mg_constrained_dofs);
    transfer.build(dof_handler);

    *pcout << "MG transfer setup time: " << time.wall_time() << "s"
           << std::endl;
  }

  template <int dim, int fe_degree>
  template <PSMF::LaplaceVariant  laplace,
            PSMF::LaplaceVariant  smooth_vmult,
            PSMF::SmootherVariant smooth_inv>
  void
  LaplaceProblem<dim, fe_degree>::do_solve(unsigned int k,
                                           unsigned int j,
                                           unsigned int i,
                                           unsigned int call_count)
  {
    // PSMF::MultigridSolver<dim,
    //                       fe_degree,
    //                       CT::DOF_LAYOUT_,
    //                       full_number,
    //                       laplace,
    //                       smooth_vmult,
    //                       smooth_inv,
    //                       vcycle_number>
    //   solver(dof_handler,
    //          mfdata,
    //          level_mfdata,
    //          edge_up_mfdata,
    //          edge_down_mfdata,
    //          patch_data_dp,
    //          patch_data_sp,
    //          transfer,
    //          Solution<dim, full_number>(),
    //          RightHandSide<dim, full_number>(),
    //          pcout,
    //          1);


    PSMF::MultigridSolverChebyshev<dim, fe_degree, CT::DOF_LAYOUT_, full_number>
      solver(dof_handler,
             mfdata,
             level_mfdata,
             edge_up_mfdata,
             edge_down_mfdata,
             transfer,
             Functions::SlitSingularityFunction<dim>(),
             Functions::ZeroFunction<dim, full_number>(),
             pcout,
             1);

    *pcout << "\nMG with [" << LaplaceToString(CT::LAPLACE_TYPE_[k]) << " "
           << LaplaceToString(CT::SMOOTH_VMULT_[j]) << " "
           << SmootherToString(CT::SMOOTH_INV_[i]) << "]\n";

    unsigned int index =
      (k * CT::SMOOTH_VMULT_.size() + j) * CT::SMOOTH_INV_.size() + i;

    info_table[index].add_value("level", triangulation.n_global_levels());
    info_table[index].add_value("cells", triangulation.n_global_active_cells());
    info_table[index].add_value("dofs", dof_handler.n_dofs());

    std::vector<PSMF::SolverData> comp_data = solver.static_comp();
    for (auto &data : comp_data)
      {
        *pcout << data.print_comp();

        auto times = data.solver_name + "[s]";
        auto perfs = data.solver_name + "Perf[Dof/s]";

        info_table[index].add_value(times, data.timing);
        info_table[index].add_value(perfs, data.perf);

        if (call_count == 0)
          {
            info_table[index].set_scientific(times, true);
            info_table[index].set_precision(times, 3);
            info_table[index].set_scientific(perfs, true);
            info_table[index].set_precision(perfs, 3);

            info_table[index].add_column_to_supercolumn(times,
                                                        data.solver_name);
            info_table[index].add_column_to_supercolumn(perfs,
                                                        data.solver_name);
          }
      }

    *pcout << std::endl;

    std::vector<PSMF::SolverData> solver_data = solver.solve();
    for (auto &data : solver_data)
      {
        *pcout << data.print_solver();

        auto it    = data.solver_name + "it";
        auto step  = data.solver_name + "step";
        auto times = data.solver_name + "[s]";
        auto perf  = data.solver_name + "[s/Dof]";
        auto mem   = data.solver_name + "Mem Usage[MB]";

        info_table[index].add_value(it, data.n_iteration);
        info_table[index].add_value(step, data.n_step);
        info_table[index].add_value(times, data.timing);
        info_table[index].add_value(perf, data.timing / dof_handler.n_dofs());
        info_table[index].add_value(mem, data.mem_usage);

        if (call_count == 0)
          {
            info_table[index].set_scientific(times, true);
            info_table[index].set_precision(times, 3);

            info_table[index].set_scientific(perf, true);
            info_table[index].set_precision(perf, 3);

            info_table[index].add_column_to_supercolumn(it, data.solver_name);
            info_table[index].add_column_to_supercolumn(step, data.solver_name);
            info_table[index].add_column_to_supercolumn(times,
                                                        data.solver_name);
            info_table[index].add_column_to_supercolumn(perf, data.solver_name);
            info_table[index].add_column_to_supercolumn(mem, data.solver_name);
          }
      }

    if (CT::SETS_ == "error_analysis")
      {
        auto solution = solver.get_solution();

        LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
                                               solution_host(solution.size());
        LinearAlgebra::ReadWriteVector<double> rw_vector(solution.size());
        rw_vector.import(solution, VectorOperation::insert);
        solution_host.import(rw_vector, VectorOperation::insert);
        ghost_solution_host = solution_host;
        constraints.distribute(ghost_solution_host);

        auto estimated = solver.get_estimate();
        LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
                                               estimate_host(estimated.size());
        LinearAlgebra::ReadWriteVector<double> rw_vector_estimate(
          estimated.size());
        rw_vector_estimate.import(estimated, VectorOperation::insert);
        estimate_host.import(rw_vector_estimate, VectorOperation::insert);

        estimated_error_per_cell.reinit(estimate_host.size());
        for (unsigned int i = 0; i < estimate_host.size(); ++i)
          estimated_error_per_cell[i] = std::sqrt(estimate_host[i]);

        const auto [l2_error, H1_error] = compute_error();

        *pcout << "L2 error: " << l2_error << std::endl
               << "H1 error: " << H1_error << std::endl
               << std::endl;

        // ghost_solution_host.print(std::cout);

        info_table[index].add_value("L2_error", l2_error);
        info_table[index].set_scientific("L2_error", true);
        info_table[index].set_precision("L2_error", 3);

        info_table[index].evaluate_convergence_rates(
          "L2_error", "dofs", ConvergenceTable::reduction_rate_log2, dim);

        info_table[index].add_value("H1_error", H1_error);
        info_table[index].set_scientific("H1_error", true);
        info_table[index].set_precision("H1_error", 3);

        info_table[index].evaluate_convergence_rates(
          "H1_error", "dofs", ConvergenceTable::reduction_rate_log2, dim);
      }
  }

  template <int dim, int fe_degree>
  std::pair<double, double>
  LaplaceProblem<dim, fe_degree>::compute_error()
  {
    Vector<double> cellwise_norm(triangulation.n_active_cells());
    VectorTools::integrate_difference(dof_handler,
                                      ghost_solution_host,
                                      Functions::SlitSingularityFunction<dim>(),
                                      cellwise_norm,
                                      QGauss<dim>(fe->degree + 1),
                                      VectorTools::L2_norm);
    const double global_norm =
      VectorTools::compute_global_error(triangulation,
                                        cellwise_norm,
                                        VectorTools::L2_norm);

    Vector<double> cellwise_h1norm(triangulation.n_active_cells());
    VectorTools::integrate_difference(dof_handler,
                                      ghost_solution_host,
                                      Functions::SlitSingularityFunction<dim>(),
                                      cellwise_h1norm,
                                      QGauss<dim>(fe->degree + 1),
                                      VectorTools::H1_seminorm);
    const double global_h1norm =
      VectorTools::compute_global_error(triangulation,
                                        cellwise_h1norm,
                                        VectorTools::H1_seminorm);

    return std::make_pair(global_norm, global_h1norm);
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::solve_mg(unsigned int n_mg_cycles)
  {
    static unsigned int call_count = 0;

    using LA = PSMF::LaplaceVariant;
    using SM = PSMF::SmootherVariant;

    // do_solve<CT::LAPLACE_TYPE_[0], CT::SMOOTH_VMULT_[0], CT::SMOOTH_INV_[0]>(
    //   0, 0, 0, call_count);

    for (unsigned int k = 0; k < CT::SMOOTH_INV_.size(); ++k)
      {
        switch (CT::SMOOTH_INV_[k])
          {
            case PSMF::SmootherVariant::GLOBAL:
              {
                do_solve<CT::LAPLACE_TYPE_[0],
                         CT::SMOOTH_VMULT_[0],
                         PSMF::SmootherVariant::GLOBAL>(0, 0, k, call_count);
                break;
              }
            case PSMF::SmootherVariant::FUSED_L:
              {
                do_solve<CT::LAPLACE_TYPE_[0],
                         CT::SMOOTH_VMULT_[0],
                         PSMF::SmootherVariant::FUSED_L>(0, 0, k, call_count);
                break;
              }
            case PSMF::SmootherVariant::ConflictFree:
              {
                do_solve<CT::LAPLACE_TYPE_[0],
                         CT::SMOOTH_VMULT_[0],
                         PSMF::SmootherVariant::ConflictFree>(0,
                                                              0,
                                                              k,
                                                              call_count);
                break;
              }
            case PSMF::SmootherVariant::ExactRes:
              {
                do_solve<CT::LAPLACE_TYPE_[0],
                         CT::SMOOTH_VMULT_[0],
                         PSMF::SmootherVariant::ExactRes>(0, 0, k, call_count);
                break;
              }
            default:
              AssertThrow(false, ExcMessage("Invalid Smoother Variant."));
          }
      }
    // for (unsigned int k = 0; k < CT::LAPLACE_TYPE_.size(); ++k)
    //   for (unsigned int j = 0; j < CT::SMOOTH_VMULT_.size(); ++j)
    //     for (unsigned int i = 0; i < CT::SMOOTH_INV_.size(); ++i)
    //       {
    //         if (LAPLACE_TYPE_[i] == LA::Basic)

    //       }



    call_count++;
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::run(const unsigned int n_cycles)
  {
    *pcout << Util::generic_info_to_fstring() << std::endl;

    for (unsigned int cycle = 0; cycle < n_cycles; ++cycle)
      {
        *pcout << "Cycle " << cycle << std::endl;

        long long unsigned int n_dofs = std::pow(
          std::pow(2, triangulation.n_global_levels()) * (fe_degree + 1), dim);

        if (n_dofs > CT::MAX_SIZES_ || cycle == n_cycles - 1)
          {
            *pcout << "Max size reached, terminating." << std::endl;
            *pcout << std::endl;

            for (unsigned int k = 0; k < CT::LAPLACE_TYPE_.size(); ++k)
              for (unsigned int j = 0; j < CT::SMOOTH_VMULT_.size(); ++j)
                for (unsigned int i = 0; i < CT::SMOOTH_INV_.size(); ++i)
                  {
                    unsigned int index = (k * CT::SMOOTH_VMULT_.size() + j) *
                                           CT::SMOOTH_INV_.size() +
                                         i;

                    std::ostringstream oss;

                    oss << "\n[" << LaplaceToString(CT::LAPLACE_TYPE_[k]) << " "
                        << LaplaceToString(CT::SMOOTH_VMULT_[j]) << " "
                        << SmootherToString(CT::SMOOTH_INV_[i]) << "]\n";
                    info_table[index].write_text(oss);

                    *pcout << oss.str() << std::endl;
                  }

            return;
          }

        if (cycle == 0)
          {
            // auto n_replicate =
            //   Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);

            parallel::distributed::Triangulation<dim> tria(
              MPI_COMM_WORLD,
              Triangulation<dim>::limit_level_difference_at_vertices,
              parallel::distributed::Triangulation<
                dim>::construct_multigrid_hierarchy);

            // GridGenerator::hyper_cube(tria, 0, 1);
            // if (dim == 2)
            //   GridGenerator::replicate_triangulation(tria,
            //                                          {CT::N_REPLICATE_, 1},
            //                                          triangulation);
            // else if (dim == 3)
            //   GridGenerator::replicate_triangulation(tria,
            //                                          {CT::N_REPLICATE_, 1,
            //                                          1}, triangulation);

            GridGenerator::hyper_cube_slit(triangulation, -1, 1);

            // SphericalManifold<dim>                boundary_manifold;
            // TransfiniteInterpolationManifold<dim> inner_manifold;

            // GridGenerator::hyper_ball(triangulation);

            // triangulation.set_all_manifold_ids(1);
            // triangulation.set_all_manifold_ids_on_boundary(0);

            // triangulation.set_manifold(0, boundary_manifold);

            // inner_manifold.initialize(triangulation);
            // triangulation.set_manifold(1, inner_manifold);
            triangulation.refine_global(1);

            // auto begin_cell = triangulation.begin_active();
            // // begin_cell->set_refine_flag();
            // begin_cell++;
            // // begin_cell->set_refine_flag();
            // begin_cell++;
            // // begin_cell->set_refine_flag();
            // begin_cell++;
            // begin_cell->set_refine_flag();
            // triangulation.execute_coarsening_and_refinement();
          }
        else
          {
            // global
            // triangulation.refine_global(1);

            // for (auto &cell : triangulation.active_cell_iterators())
            //   {
            //     // quad
            //     auto center = cell->center();
            //     if (dim == 2)
            //       {
            //         if (center[0] > 0.5 && center[1] > 0.5)
            //           cell->set_refine_flag();
            //       }
            //     else if (dim == 3)
            //       {
            //         if (center[0] > 0.5 && center[1] > 0.5 && center[2] >
            //         0.5)
            //           cell->set_refine_flag();
            //       }
            //   }

            //     // // circle
            //     // const Point<dim> center;
            //     // const double     radius = 1. / 2;
            //     // for (const auto v : cell->vertex_indices())
            //     //   {
            //     //     auto distance_from_center =
            //     //     center.distance(cell->vertex(v));

            //     //     if (distance_from_center < radius)
            //     //       {
            //     //         cell->set_refine_flag();
            //     //         break;
            //     //       }
            //     //   }
            //   }
            parallel::distributed::GridRefinement::
              refine_and_coarsen_fixed_fraction(triangulation,
                                                estimated_error_per_cell,
                                                0.5,
                                                0.0);
            triangulation.execute_coarsening_and_refinement();

            // estimated_error_per_cell.print(std::cout);
            // ghost_solution_host.print(std::cout);
          }

        setup_system();
        assemble_mg();

        solve_mg(1);
        *pcout << std::endl;
      }
  }
} // namespace Step64
int
main(int argc, char *argv[])
{
  try
    {
      using namespace Step64;

      Utilities::MPI::MPI_InitFinalize mpi_init(argc, argv, 1);

      {
        int         n_devices       = 0;
        hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
        AssertCuda(cuda_error_code);
        const unsigned int my_mpi_id =
          Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
        const int device_id = my_mpi_id % n_devices;
        cuda_error_code     = hipSetDevice(device_id);
        AssertCuda(cuda_error_code);
      }

      {
        LaplaceProblem<CT::DIMENSION_, CT::FE_DEGREE_> Laplace_problem;
        Laplace_problem.run(10);
      }
    }
  catch (std::exception &exc)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Exception on processing: " << std::endl
                << exc.what() << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  catch (...)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Unknown exception!" << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  return 0;
}