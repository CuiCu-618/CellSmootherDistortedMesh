#include "hip/hip_runtime.h"
/**
 * Created by Cu Cui on 2022/12/25.
 */

#include <deal.II/base/conditional_ostream.h>
#include <deal.II/base/convergence_table.h>
#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/function.h>
#include <deal.II/base/quadrature_lib.h>
#include <deal.II/base/timer.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_q.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/tria.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/la_parallel_vector.h>

#include <deal.II/numerics/data_out.h>
#include <deal.II/numerics/vector_tools.h>

#include <hip/hip_runtime_api.h>

#include <fstream>

#include "app_utilities.h"
#include "ct_parameter.h"
#include "solver.cuh"
#include "utilities.cuh"


// -\delta u = f, u = 0 on \parital \Omege, f = 1.
// double percision

namespace Step64
{
  using namespace dealii;

  template <int dim, typename Number>
  class Solution : public Function<dim, Number>
  {
  public:
    virtual Number
    value(const Point<dim> &, const unsigned int = 0) const override final
    {
      return 0.;
    }

    virtual Tensor<1, dim, Number>
    gradient(const Point<dim> &p, const unsigned int = 0) const override final
    {
      Tensor<1, dim, Number> grad;
      for (unsigned int d = 0; d < dim; ++d)
        {
          grad[d] = 1.;
          for (unsigned int e = 0; e < dim; ++e)
            if (d == e)
              grad[d] *= -numbers::PI * std::cos(numbers::PI * p[e]);
            else
              grad[d] *= std::sin(numbers::PI * p[e]);
        }
      return grad;
    }
  };

  template <int dim, typename Number>
  class RightHandSide : public Function<dim, Number>
  {
  public:
    virtual Number
    value(const Point<dim> &, const unsigned int = 0) const override final
    {
      return 1.;
    }
  };

  template <int dim, int fe_degree>
  class LaplaceProblem
  {
  public:
    using full_number = double;
    using MatrixTypeDP =
      PSMF::LaplaceOperator<dim, fe_degree, full_number, CT::DOF_LAYOUT_>;

    LaplaceProblem();
    ~LaplaceProblem();
    void
    run(const unsigned int n_cycles);

  private:
    void
    setup_system();
    void
    assemble_mg();
    void
    solve_mg(unsigned int n_mg_cycles);

    Triangulation<dim> triangulation;
    FE_Q<dim>          fe;
    DoFHandler<dim>    dof_handler;
    MappingQ1<dim>     mapping;
    double             setup_time;
    ConvergenceTable   convergence_table;

    std::fstream                        fout;
    std::shared_ptr<ConditionalOStream> pcout;

    MGLevelObject<MatrixTypeDP>                             matrix_dp;
    PSMF::MGTransferCUDA<dim, full_number, CT::DOF_LAYOUT_> transfer;
  };

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::LaplaceProblem()
    : triangulation(Triangulation<dim>::limit_level_difference_at_vertices)
    , fe(fe_degree)
    , dof_handler(triangulation)
    , setup_time(0.)
    , pcout(std::make_shared<ConditionalOStream>(std::cout, false))
  {
    const auto filename = Util::get_filename();
    fout.open(filename + ".log", std::ios_base::out);
    pcout = std::make_shared<ConditionalOStream>(fout, true);
  }

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::~LaplaceProblem()
  {
    fout.close();
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::setup_system()
  {
    Timer time;
    setup_time = 0;

    dof_handler.distribute_dofs(fe);
    dof_handler.distribute_mg_dofs();
    const unsigned int nlevels = triangulation.n_global_levels();
    for (unsigned int level = 0; level < nlevels; ++level)
      Util::Lexicographic(dof_handler, level);
    Util::Lexicographic(dof_handler);

    *pcout << "Number of degrees of freedom: " << dof_handler.n_dofs() << " = ("
           << ((int)std::pow(dof_handler.n_dofs() * 1.0000001, 1. / dim) - 1) /
                fe.degree
           << " x " << fe.degree << " + 1)^" << dim << std::endl;

    setup_time += time.wall_time();

    *pcout << "DoF setup time:         " << setup_time << "s" << std::endl;
  }
  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::assemble_mg()
  {
    MGConstrainedDoFs mg_constrained_dofs;

    // Initialization of Dirichlet boundaries
    std::set<types::boundary_id> dirichlet_boundary;
    dirichlet_boundary.insert(0);
    mg_constrained_dofs.initialize(dof_handler);
    mg_constrained_dofs.make_zero_boundary_constraints(dof_handler,
                                                       dirichlet_boundary);

    // set up a mapping for the geometry representation
    MappingQ1<dim> mapping;

    unsigned int minlevel = 1;
    unsigned int maxlevel = triangulation.n_global_levels() - 1;

    matrix_dp.resize(1, maxlevel);

    Timer time;
    for (unsigned int level = minlevel; level <= maxlevel; ++level)
      {
        IndexSet relevant_dofs;
        DoFTools::extract_locally_relevant_level_dofs(dof_handler,
                                                      level,
                                                      relevant_dofs);

        AffineConstraints<full_number> level_constraints;
        level_constraints.reinit(relevant_dofs);
        level_constraints.add_lines(
          mg_constrained_dofs.get_boundary_indices(level));
        level_constraints.close();

        // double-precision matrix-free data
        {
          using MatrixFreeType = PSMF::MatrixFree<dim, full_number>;

          typename MatrixFreeType::AdditionalData additional_data;
          additional_data.mapping_update_flags =
            (update_values | update_gradients | update_JxW_values);
          additional_data.mg_level = level;
          std::shared_ptr<MatrixFreeType> mg_mf_storage_level(
            new MatrixFreeType());
          mg_mf_storage_level->reinit(mapping,
                                      dof_handler,
                                      level_constraints,
                                      QGauss<1>(fe_degree + 1),
                                      additional_data);

          matrix_dp[level].initialize(mg_mf_storage_level);
        }
      }

    *pcout << "Matrix-free setup time: " << time.wall_time() << "s"
           << std::endl;

    time.restart();
    transfer.initialize_constraints(mg_constrained_dofs);
    transfer.build(dof_handler);

    *pcout << "MG transfer setup time: " << time.wall_time() << "s"
           << std::endl;
  }
  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::solve_mg(unsigned int n_mg_cycles)
  {
    auto do_solve = [&]()
    {};
    
    PSMF::MultigridSolver<dim,
                          fe_degree,
                          CT::DOF_LAYOUT_,
                          double,
                          CT::KERNEL_TYPE_,
                          CT::VCYCLE_NUMBER_>
      solver(dof_handler,
             matrix_dp,
             transfer,
             Functions::ZeroFunction<dim, double>(),
             Functions::ConstantFunction<dim, double>(1.),
             pcout,
             n_mg_cycles);


    *pcout << std::endl;

    convergence_table.add_value("level", triangulation.n_global_levels());
    convergence_table.add_value("cells", triangulation.n_global_active_cells());
    convergence_table.add_value("dofs", dof_handler.n_dofs());

    static unsigned int call_count = 0;

    std::vector<PSMF::SolverData> comp_data = solver.static_comp();
    for (auto &data : comp_data)
      {
        *pcout << data.print_comp();

        auto times = data.solver_name + "[s]";
        auto perfs = data.solver_name + "Perf[Dof/s]";

        convergence_table.add_value(times, data.timing);
        convergence_table.add_value(perfs, data.perf);

        if (call_count == 0)
          {
            convergence_table.set_scientific(times, true);
            convergence_table.set_precision(times, 3);
            convergence_table.set_scientific(perfs, true);
            convergence_table.set_precision(perfs, 3);

            convergence_table.add_column_to_supercolumn(times,
                                                        data.solver_name);
            convergence_table.add_column_to_supercolumn(perfs,
                                                        data.solver_name);
          }
      }

    *pcout << std::endl;

    std::vector<PSMF::SolverData> solver_data = solver.solve();
    for (auto &data : solver_data)
      {
        *pcout << data.print_solver();

        auto it    = data.solver_name + "it";
        auto times = data.solver_name + "[s]";
        auto mem   = data.solver_name + "Mem Usage[MB]";

        convergence_table.add_value(it, data.n_iteration);
        convergence_table.add_value(times, data.timing);
        convergence_table.add_value(mem, data.mem_usage);

        if (call_count == 0)
          {
            convergence_table.set_scientific(times, true);
            convergence_table.set_precision(times, 3);

            convergence_table.add_column_to_supercolumn(it, data.solver_name);
            convergence_table.add_column_to_supercolumn(times,
                                                        data.solver_name);
            convergence_table.add_column_to_supercolumn(mem, data.solver_name);
          }
      }

    call_count++;
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::run(const unsigned int n_cycles)
  {
    *pcout << Util::generic_info_to_fstring() << std::endl;

    for (unsigned int cycle = 0; cycle < n_cycles; ++cycle)
      {
        smoother_mem = 0;

        *pcout << "Cycle " << cycle << std::endl;

        long long unsigned int n_dofs =
          std::pow(std::pow(2, triangulation.n_global_levels()) * fe_degree + 1,
                   dim);

        if (n_dofs > CT::MAX_SIZES_)
          {
            *pcout << "Max size reached, terminating." << std::endl;
            *pcout << std::endl;

            std::ostringstream oss;
            convergence_table.write_text(oss);
            *pcout << oss.str() << std::endl;

            return;
          }

        if (cycle == 0)
          {
            GridGenerator::hyper_cube(triangulation, 0., 1.);
            triangulation.refine_global(2);
          }
        else
          triangulation.refine_global(1);

        setup_system();
        assemble_mg();

        solve_mg(1);
        *pcout << std::endl;
      }
  }
} // namespace Step64
int
main(int argc, char *argv[])
{
  try
    {
      using namespace Step64;

      {
        int device_id = findCudaDevice(argc, (const char **)argv);
        AssertCuda(hipSetDevice(device_id));
      }

      {
        LaplaceProblem<CT::DIMENSION_, CT::FE_DEGREE_> Laplace_problem;
        Laplace_problem.run(20);
      }
    }
  catch (std::exception &exc)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Exception on processing: " << std::endl
                << exc.what() << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  catch (...)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Unknown exception!" << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  return 0;
}