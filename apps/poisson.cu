#include "hip/hip_runtime.h"
/**
 * @file poisson.cu
 * @author Cu Cui (cu.cui@iwr.uni-heidelberg.de)
 * @brief Discontinuous Galerkin methods for poisson problems.
 * @version 1.0
 * @date 2023-02-02
 *
 * @copyright Copyright (c) 2023
 *
 */

#include <deal.II/base/conditional_ostream.h>
#include <deal.II/base/convergence_table.h>
#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/function.h>
#include <deal.II/base/quadrature_lib.h>
#include <deal.II/base/timer.h>

#include <deal.II/distributed/tria.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_dgq.h>
#include <deal.II/fe/fe_q.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/grid_tools.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/la_parallel_vector.h>

#include <deal.II/numerics/data_out.h>
#include <deal.II/numerics/vector_tools.h>

#include <hip/hip_runtime_api.h>

#include <fstream>

#include "app_utilities.h"
#include "ct_parameter.h"
#include "solver.cuh"
#include "utilities.cuh"


// -\delta u = f, u = 0 on \parital \Omege, f = 1.
// double percision

namespace Step64
{
  using namespace dealii;

  template <int dim, typename Number>
  class Solution : public Function<dim, Number>
  {
  public:
    virtual Number
    value(const Point<dim> &p, const unsigned int = 0) const override final
    {
      Number val = 1.;
      for (unsigned int d = 0; d < dim; ++d)
        val *= std::sin(numbers::PI * p[d]);
      return val;
    }

    virtual Tensor<1, dim, Number>
    gradient(const Point<dim> &p, const unsigned int = 0) const override final
    {
      Tensor<1, dim, Number> grad;
      for (unsigned int d = 0; d < dim; ++d)
        {
          grad[d] = 1.;
          for (unsigned int e = 0; e < dim; ++e)
            if (d == e)
              grad[d] *= -numbers::PI * std::cos(numbers::PI * p[e]);
            else
              grad[d] *= std::sin(numbers::PI * p[e]);
        }
      return grad;
    }
  };

  template <int dim, typename Number>
  class RightHandSide : public Function<dim, Number>
  {
  public:
    virtual Number
    value(const Point<dim> &p, const unsigned int = 0) const override final
    {
      const Number arg = numbers::PI;
      Number       val = 1.;
      for (unsigned int d = 0; d < dim; ++d)
        val *= std::sin(arg * p[d]);
      return dim * arg * arg * val;
    }
  };

  template <int dim, int fe_degree>
  class LaplaceProblem
  {
  public:
    using full_number   = double;
    using vcycle_number = CT::VCYCLE_NUMBER_;
    using MatrixFreeDP  = PSMF::LevelVertexPatch<dim, fe_degree, full_number>;
    using MatrixFreeSP  = PSMF::LevelVertexPatch<dim, fe_degree, vcycle_number>;

    LaplaceProblem();
    ~LaplaceProblem();
    void
    run(const unsigned int n_cycles);

  private:
    void
    setup_system();
    void
    assemble_mg();
    void
    solve_mg(unsigned int n_mg_cycles);

    template <PSMF::LaplaceVariant  laplace,
              PSMF::LaplaceVariant  smooth_vmult,
              PSMF::SmootherVariant smooth_inv>
    void
    do_solve(unsigned int k,
             unsigned int j,
             unsigned int i,
             unsigned int call_count);

    MPI_Comm                                  mpi_communicator;
    parallel::distributed::Triangulation<dim> triangulation;
    std::shared_ptr<FiniteElement<dim>>       fe;
    DoFHandler<dim>                           dof_handler;
    MappingQ1<dim>                            mapping;
    double                                    setup_time;

    std::vector<ConvergenceTable> info_table;

    std::fstream                        fout;
    std::shared_ptr<ConditionalOStream> pcout;

    MGLevelObject<std::shared_ptr<MatrixFreeDP>> mfdata_dp;
    MGLevelObject<std::shared_ptr<MatrixFreeSP>> mfdata_sp;
    MGConstrainedDoFs                            mg_constrained_dofs;

    PSMF::MGTransferCUDA<dim, vcycle_number, CT::DOF_LAYOUT_> transfer;
  };

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::LaplaceProblem()
    : mpi_communicator(MPI_COMM_WORLD)
    , triangulation(MPI_COMM_WORLD,
                    Triangulation<dim>::limit_level_difference_at_vertices,
                    parallel::distributed::Triangulation<
                      dim>::construct_multigrid_hierarchy)
    , fe([&]() -> std::shared_ptr<FiniteElement<dim>> {
      if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::Q)
        return std::make_shared<FE_Q<dim>>(fe_degree);
      else if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::DGQ)
        return std::make_shared<FE_DGQHermite<dim>>(fe_degree);
      return std::shared_ptr<FiniteElement<dim>>();
    }())
    , dof_handler(triangulation)
    , setup_time(0.)
    , pcout(std::make_shared<ConditionalOStream>(std::cout, false))
  {
    const auto filename = Util::get_filename();
    if (Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
      {
        fout.open(filename + ".log", std::ios_base::out);
        pcout = std::make_shared<ConditionalOStream>(
          fout, Utilities::MPI::this_mpi_process(mpi_communicator) == 0);
      }

    info_table.resize(CT::LAPLACE_TYPE_.size() * CT::SMOOTH_VMULT_.size() *
                      CT::SMOOTH_INV_.size());
  }

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::~LaplaceProblem()
  {
    if (Utilities::MPI::this_mpi_process(mpi_communicator) == 0)
      fout.close();
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::setup_system()
  {
    Timer time;
    setup_time = 0;

    dof_handler.distribute_dofs(*fe);
    dof_handler.distribute_mg_dofs();
    const unsigned int nlevels = triangulation.n_global_levels();

    auto n_replicate =
      CT::IS_REPLICATE_ ? Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD) : 1;

    *pcout << "Number of degrees of freedom: " << dof_handler.n_dofs() << " = "
           << n_replicate << " x (" << (1 << (nlevels - 1)) << " x ("
           << fe->degree << " + 1))^" << dim << std::endl;

    setup_time += time.wall_time();

    *pcout << "DoF setup time:         " << setup_time << "s" << std::endl;
  }
  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::assemble_mg()
  {
    // Initialization of Dirichlet boundaries
    std::set<types::boundary_id> dirichlet_boundary;
    dirichlet_boundary.insert(0);
    mg_constrained_dofs.initialize(dof_handler);
    mg_constrained_dofs.make_zero_boundary_constraints(dof_handler,
                                                       dirichlet_boundary);

    // set up a mapping for the geometry representation
    MappingQ1<dim> mapping;

    unsigned int minlevel = 1;
    unsigned int maxlevel = triangulation.n_global_levels() - 1;

    mfdata_dp.resize(1, maxlevel);

    if (std::is_same_v<vcycle_number, float>)
      mfdata_sp.resize(1, maxlevel);

    Timer time;
    for (unsigned int level = minlevel; level <= maxlevel; ++level)
      {
        // IndexSet relevant_dofs;
        // DoFTools::extract_locally_relevant_level_dofs(dof_handler,
        //                                               level,
        //                                               relevant_dofs);
        // double-precision matrix-free data
        {
          // AffineConstraints<full_number> level_constraints;
          // level_constraints.reinit(relevant_dofs);
          // level_constraints.add_lines(
          //   mg_constrained_dofs.get_boundary_indices(level));
          // level_constraints.close();

          typename MatrixFreeDP::AdditionalData additional_data;
          additional_data.relaxation         = 1.;
          additional_data.use_coloring       = false;
          additional_data.patch_per_block    = CT::PATCH_PER_BLOCK_;
          additional_data.granularity_scheme = CT::GRANULARITY_;

          mfdata_dp[level] = std::make_shared<MatrixFreeDP>();
          mfdata_dp[level]->reinit(dof_handler, level, additional_data);
        }

        // single-precision matrix-free data
        if (std::is_same_v<vcycle_number, float>)
          {
            // AffineConstraints<vcycle_number> level_constraints;
            // level_constraints.reinit(relevant_dofs);
            // level_constraints.add_lines(
            //   mg_constrained_dofs.get_boundary_indices(level));
            // level_constraints.close();

            typename MatrixFreeSP::AdditionalData additional_data;
            additional_data.relaxation         = 1.;
            additional_data.use_coloring       = false;
            additional_data.patch_per_block    = CT::PATCH_PER_BLOCK_;
            additional_data.granularity_scheme = CT::GRANULARITY_;

            mfdata_sp[level] = std::make_shared<MatrixFreeSP>();
            mfdata_sp[level]->reinit(dof_handler, level, additional_data);
          }
      }

    *pcout << "Matrix-free setup time: " << time.wall_time() << "s"
           << std::endl;

    time.restart();

    transfer.initialize_constraints(mg_constrained_dofs);
    transfer.build(dof_handler);

    *pcout << "MG transfer setup time: " << time.wall_time() << "s"
           << std::endl;
  }

  template <int dim, int fe_degree>
  template <PSMF::LaplaceVariant  laplace,
            PSMF::LaplaceVariant  smooth_vmult,
            PSMF::SmootherVariant smooth_inv>
  void
  LaplaceProblem<dim, fe_degree>::do_solve(unsigned int k,
                                           unsigned int j,
                                           unsigned int i,
                                           unsigned int call_count)
  {
    PSMF::MultigridSolver<dim,
                          fe_degree,
                          CT::DOF_LAYOUT_,
                          full_number,
                          laplace,
                          smooth_vmult,
                          smooth_inv,
                          vcycle_number>
      solver(dof_handler,
             mfdata_dp,
             mfdata_sp,
             transfer,
             Functions::ZeroFunction<dim, full_number>(),
             Functions::ConstantFunction<dim, full_number>(1.),
             pcout,
             1);

    *pcout << "\nMG with [" << LaplaceToString(CT::LAPLACE_TYPE_[k]) << " "
           << LaplaceToString(CT::SMOOTH_VMULT_[j]) << " "
           << SmootherToString(CT::SMOOTH_INV_[i]) << "]\n";

    unsigned int index =
      (k * CT::SMOOTH_VMULT_.size() + j) * CT::SMOOTH_INV_.size() + i;

    info_table[index].add_value("level", triangulation.n_global_levels());
    info_table[index].add_value("cells", triangulation.n_global_active_cells());
    info_table[index].add_value("dofs", dof_handler.n_dofs());

    std::vector<PSMF::SolverData> comp_data = solver.static_comp();
    for (auto &data : comp_data)
      {
        *pcout << data.print_comp();

        auto times = data.solver_name + "[s]";
        auto perfs = data.solver_name + "Perf[Dof/s]";

        info_table[index].add_value(times, data.timing);
        info_table[index].add_value(perfs, data.perf);

        if (call_count == 0)
          {
            info_table[index].set_scientific(times, true);
            info_table[index].set_precision(times, 3);
            info_table[index].set_scientific(perfs, true);
            info_table[index].set_precision(perfs, 3);

            info_table[index].add_column_to_supercolumn(times,
                                                        data.solver_name);
            info_table[index].add_column_to_supercolumn(perfs,
                                                        data.solver_name);
          }
      }

    *pcout << std::endl;

    std::vector<PSMF::SolverData> solver_data = solver.solve();
    for (auto &data : solver_data)
      {
        *pcout << data.print_solver();

        auto it    = data.solver_name + "it";
        auto step  = data.solver_name + "step";
        auto times = data.solver_name + "[s]";
        auto perf  = data.solver_name + "[s/Dof]";
        auto mem   = data.solver_name + "Mem Usage[MB]";

        info_table[index].add_value(it, data.n_iteration);
        info_table[index].add_value(step, data.n_step);
        info_table[index].add_value(times, data.timing);
        info_table[index].add_value(perf, data.timing / dof_handler.n_dofs());
        info_table[index].add_value(mem, data.mem_usage);

        if (call_count == 0)
          {
            info_table[index].set_scientific(times, true);
            info_table[index].set_precision(times, 3);

            info_table[index].set_scientific(perf, true);
            info_table[index].set_precision(perf, 3);

            info_table[index].add_column_to_supercolumn(it, data.solver_name);
            info_table[index].add_column_to_supercolumn(step, data.solver_name);
            info_table[index].add_column_to_supercolumn(times,
                                                        data.solver_name);
            info_table[index].add_column_to_supercolumn(perf, data.solver_name);
            info_table[index].add_column_to_supercolumn(mem, data.solver_name);
          }
      }
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::solve_mg(unsigned int n_mg_cycles)
  {
    static unsigned int call_count = 0;

    using LA = PSMF::LaplaceVariant;
    using SM = PSMF::SmootherVariant;

    do_solve<CT::LAPLACE_TYPE_[0], CT::SMOOTH_VMULT_[0], CT::SMOOTH_INV_[0]>(
      0, 0, 0, call_count);

    // for (unsigned int k = 0; k < CT::LAPLACE_TYPE_.size(); ++k)
    //   for (unsigned int j = 0; j < CT::SMOOTH_VMULT_.size(); ++j)
    //     for (unsigned int i = 0; i < CT::SMOOTH_INV_.size(); ++i)
    //       {
    //         if (LAPLACE_TYPE_[i] == LA::Basic)

    //       }



    call_count++;
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::run(const unsigned int n_cycles)
  {
    *pcout << Util::generic_info_to_fstring() << std::endl;

    for (unsigned int cycle = 0; cycle < n_cycles; ++cycle)
      {
        *pcout << "Cycle " << cycle << std::endl;

        long long unsigned int n_dofs = std::pow(
          std::pow(2, triangulation.n_global_levels()) * (fe_degree + 1), dim);

        if (n_dofs > CT::MAX_SIZES_)
          {
            *pcout << "Max size reached, terminating." << std::endl;
            *pcout << std::endl;

            for (unsigned int k = 0; k < CT::LAPLACE_TYPE_.size(); ++k)
              for (unsigned int j = 0; j < CT::SMOOTH_VMULT_.size(); ++j)
                for (unsigned int i = 0; i < CT::SMOOTH_INV_.size(); ++i)
                  {
                    unsigned int index = (k * CT::SMOOTH_VMULT_.size() + j) *
                                           CT::SMOOTH_INV_.size() +
                                         i;

                    std::ostringstream oss;

                    oss << "\n[" << LaplaceToString(CT::LAPLACE_TYPE_[k]) << " "
                        << LaplaceToString(CT::SMOOTH_VMULT_[j]) << " "
                        << SmootherToString(CT::SMOOTH_INV_[i]) << "]\n";
                    info_table[index].write_text(oss);

                    *pcout << oss.str() << std::endl;
                  }

            return;
          }

        if (cycle == 0)
          {
            if (CT::IS_REPLICATE_)
              {
                auto n_replicate =
                  Utilities::MPI::n_mpi_processes(MPI_COMM_WORLD);
                Tensor<1, dim> shift_vector;
                shift_vector[0] = 1;

                parallel::distributed::Triangulation<dim> tria(
                  MPI_COMM_WORLD,
                  Triangulation<dim>::limit_level_difference_at_vertices,
                  parallel::distributed::Triangulation<
                    dim>::construct_multigrid_hierarchy);

                GridGenerator::hyper_cube(tria, 0, 1);
                if (dim == 2)
                  GridGenerator::replicate_triangulation(tria,
                                                         {n_replicate, 1},
                                                         triangulation);
                else if (dim == 3)
                  GridGenerator::replicate_triangulation(tria,
                                                         {n_replicate, 1, 1},
                                                         triangulation);
              }
            else
              GridGenerator::hyper_cube(triangulation, 0., 1.);

            triangulation.refine_global(2);
          }
        else
          triangulation.refine_global(1);

        setup_system();
        assemble_mg();

        solve_mg(1);
        *pcout << std::endl;
      }
  }
} // namespace Step64
int
main(int argc, char *argv[])
{
  try
    {
      using namespace Step64;

      Utilities::MPI::MPI_InitFinalize mpi_init(argc, argv, 1);

      {
        int         n_devices       = 0;
        hipError_t cuda_error_code = hipGetDeviceCount(&n_devices);
        AssertCuda(cuda_error_code);
        const unsigned int my_mpi_id =
          Utilities::MPI::this_mpi_process(MPI_COMM_WORLD);
        const int device_id = my_mpi_id % n_devices;
        cuda_error_code     = hipSetDevice(device_id);
        AssertCuda(cuda_error_code);
      }

      {
        LaplaceProblem<CT::DIMENSION_, CT::FE_DEGREE_> Laplace_problem;
        Laplace_problem.run(20);
      }
    }
  catch (std::exception &exc)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Exception on processing: " << std::endl
                << exc.what() << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  catch (...)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Unknown exception!" << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  return 0;
}