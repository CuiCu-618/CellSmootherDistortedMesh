#include "hip/hip_runtime.h"
/**
 * @file poisson.cu
 * @author Cu Cui (cu.cui@iwr.uni-heidelberg.de)
 * @brief Discontinuous Galerkin methods for poisson problems.
 * @version 1.0
 * @date 2023-02-02
 *
 * @copyright Copyright (c) 2023
 *
 */

#include <deal.II/base/conditional_ostream.h>
#include <deal.II/base/convergence_table.h>
#include <deal.II/base/hip/hip_runtime.h>
#include <deal.II/base/function.h>
#include <deal.II/base/quadrature_lib.h>
#include <deal.II/base/timer.h>

#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_dgq.h>
#include <deal.II/fe/fe_interface_values.h>
#include <deal.II/fe/fe_q.h>
#include <deal.II/fe/fe_raviart_thomas_new.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/tria.h>

#include <deal.II/lac/affine_constraints.h>
#include <deal.II/lac/la_parallel_vector.h>

#include <deal.II/numerics/data_out.h>
#include <deal.II/numerics/vector_tools.h>

#include <hip/hip_runtime_api.h>

#include <fstream>

#include "TPSS/move_to_deal_ii.h"
#include "app_utilities.h"
#include "ct_parameter.h"
#include "equation_data.h"
#include "solver.cuh"
#include "utilities.cuh"

// -\delta u = f, u = 0 on \parital \Omege, f = 1.
// double percision

namespace Step64
{
  static unsigned int call_count = 0;

  using namespace dealii;

  template <std::size_t I, std::size_t J, std::size_t K>
  struct Tester
  {
    template <typename T>
    static void
    run(T &t)
    {
      t.template do_solve<CT::LAPLACE_TYPE_[I],
                          CT::SMOOTH_VMULT_[I],
                          CT::SMOOTH_INV_[J],
                          CT::LOCAL_SOLVER_[K]>(I, J, K, call_count);
      if constexpr (J == 0 && K == 0)
        {
          Tester<I - 1,
                 CT::SMOOTH_INV_.size() - 1,
                 CT::LOCAL_SOLVER_.size() - 1>::run();
        }
      else if constexpr (K == 0)
        {
          Tester<I, J - 1, CT::LOCAL_SOLVER_.size() - 1>::run(t);
        }
      else
        {
          Tester<I, J, K - 1>::run(t);
        }
    }
  };

  template <std::size_t I, std::size_t J>
  struct Tester<I, J, 0>
  {
    template <typename T>
    static void
    run(T &t)
    {
      t.template do_solve<CT::LAPLACE_TYPE_[I],
                          CT::SMOOTH_VMULT_[I],
                          CT::SMOOTH_INV_[J],
                          CT::LOCAL_SOLVER_[0]>(I, J, 0, call_count);
      Tester<I, J - 1, CT::LOCAL_SOLVER_.size() - 1>::run(t);
    }
  };

  template <std::size_t I>
  struct Tester<I, 0, 0>
  {
    template <typename T>
    static void
    run(T &t)
    {
      t.template do_solve<CT::LAPLACE_TYPE_[I],
                          CT::SMOOTH_VMULT_[I],
                          CT::SMOOTH_INV_[0],
                          CT::LOCAL_SOLVER_[0]>(I, 0, 0, call_count);
      Tester<I - 1, CT::SMOOTH_INV_.size() - 1, CT::LOCAL_SOLVER_.size() - 1>::
        run(t);
    }
  };

  template <>
  struct Tester<0, 0, 0>
  {
    template <typename T>
    static void
    run(T &t)
    {
      t.template do_solve<CT::LAPLACE_TYPE_[0],
                          CT::SMOOTH_VMULT_[0],
                          CT::SMOOTH_INV_[0],
                          CT::LOCAL_SOLVER_[0]>(0, 0, 0, call_count);
    }
  };

  template <int dim>
  using Solution = Stokes::NoSlipExp::Solution<dim>;

  template <int dim>
  using SolutionVelocity = Stokes::NoSlipExp::SolutionVelocity<dim>;

  template <int dim>
  using SolutionPressure = Stokes::NoSlipExp::SolutionPressure<dim>;

  template <int dim>
  using RightHandSide = Stokes::ManufacturedLoad<dim>;

  template <int dim, int fe_degree>
  class LaplaceProblem
  {
  public:
    using full_number   = double;
    using vcycle_number = CT::VCYCLE_NUMBER_;
    using MatrixFreeDP  = PSMF::LevelVertexPatch<dim, fe_degree, full_number>;
    using MatrixFreeSP  = PSMF::LevelVertexPatch<dim, fe_degree, vcycle_number>;

    LaplaceProblem();
    ~LaplaceProblem();
    void
    run(const unsigned int n_cycles);

    template <PSMF::LaplaceVariant     laplace,
              PSMF::LaplaceVariant     smooth_vmult,
              PSMF::SmootherVariant    smooth_inv,
              PSMF::LocalSolverVariant local_solver>
    void
    do_solve(unsigned int k,
             unsigned int j,
             unsigned int i,
             unsigned int call_count);

  private:
    void
    setup_system();
    void
    assemble_rhs();
    void
    assemble_mg();
    void
    solve_mg(unsigned int n_mg_cycles);
    std::tuple<double, double, double>
    compute_error();

    Triangulation<dim>                  triangulation;
    std::shared_ptr<FiniteElement<dim>> fe;
    DoFHandler<dim>                     dof_handler;
    DoFHandler<dim>                     dof_handler_velocity;
    DoFHandler<dim>                     dof_handler_pressure;
    MappingQ1<dim>                      mapping;

    double setup_time;

    std::vector<ConvergenceTable> info_table;

    std::fstream                        fout;
    std::shared_ptr<ConditionalOStream> pcout;

    MGLevelObject<std::shared_ptr<MatrixFreeDP>> mfdata_dp;
    MGLevelObject<std::shared_ptr<MatrixFreeSP>> mfdata_sp;
    MGConstrainedDoFs                            mg_constrained_dofs;
    AffineConstraints<double>                    constraints;

    LinearAlgebra::distributed::Vector<double, MemorySpace::CUDA>
      system_rhs_dev;

    LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
      solution_velocity_host;
    LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
      solution_pressure_host;


    PSMF::MGTransferCUDA<dim, vcycle_number> transfer;
  };

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::LaplaceProblem()
    : triangulation(Triangulation<dim>::limit_level_difference_at_vertices)
    , fe([&]() -> std::shared_ptr<FiniteElement<dim>> {
      if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::Q)
        return std::make_shared<FE_Q<dim>>(fe_degree);
      else if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::DGQ)
        return std::make_shared<FE_DGQHermite<dim>>(fe_degree);
      else if (CT::DOF_LAYOUT_ == PSMF::DoFLayout::RT)
        return std::make_shared<FESystem<dim>>(FE_RaviartThomas_new<dim>(
                                                 fe_degree),
                                               1,
                                               FE_DGQLegendre<dim>(fe_degree),
                                               1);
      return std::shared_ptr<FiniteElement<dim>>();
    }())
    , dof_handler(triangulation)
    , dof_handler_velocity(triangulation)
    , dof_handler_pressure(triangulation)
    , setup_time(0.)
    , pcout(std::make_shared<ConditionalOStream>(std::cout, false))
  {
    const auto filename = Util::get_filename();
    fout.open(filename + ".log", std::ios_base::out);
    pcout = std::make_shared<ConditionalOStream>(fout, true);

    info_table.resize(CT::LAPLACE_TYPE_.size() * CT::SMOOTH_INV_.size() *
                      CT::LOCAL_SOLVER_.size());
  }

  template <int dim, int fe_degree>
  LaplaceProblem<dim, fe_degree>::~LaplaceProblem()
  {
    fout.close();
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::setup_system()
  {
    Timer time;
    setup_time = 0;

    dof_handler_velocity.distribute_dofs(fe->get_sub_fe(0, dim));
    dof_handler_velocity.distribute_mg_dofs();

    dof_handler_pressure.distribute_dofs(fe->get_sub_fe(dim, 1));
    dof_handler_pressure.distribute_mg_dofs();

    dof_handler.distribute_dofs(*fe);
    dof_handler.distribute_mg_dofs();

    *pcout << "Number of degrees of freedom: " << dof_handler.n_dofs() << " = ("
           << dof_handler_velocity.n_dofs() << " + "
           << dof_handler_pressure.n_dofs() << ")" << std::endl;

    constraints.clear();
    VectorToolsFix::project_boundary_values_div_conforming(
      dof_handler_velocity,
      0,
      SolutionVelocity<dim>(),
      0,
      constraints,
      mapping);
    constraints.close();

    setup_time += time.wall_time();

    *pcout << "DoF setup time:         " << setup_time << "s" << std::endl;
  }
  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::assemble_rhs()
  {
    const unsigned int n_dofs = dof_handler.n_dofs();
    system_rhs_dev.reinit(n_dofs);

    LinearAlgebra::ReadWriteVector<double> rw_vector(n_dofs);
    LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
      system_rhs_host(n_dofs);


    if (CT::SETS_ == "none")
      {
        for (unsigned int i = 0; i < dof_handler_velocity.n_dofs(); ++i)
          system_rhs_host[i] = 1.;
        rw_vector.import(system_rhs_host, VectorOperation::insert);
        system_rhs_dev.import(rw_vector, VectorOperation::insert);

        return;
      }
    Timer time;

    SolutionVelocity<dim> exact_solution;
    RightHandSide<dim>    rhs_function(std::make_shared<Solution<dim>>());

    AffineConstraints<double> constraints;
    constraints.clear();
    VectorToolsFix::project_boundary_values_div_conforming(dof_handler_velocity,
                                                           0,
                                                           exact_solution,
                                                           0,
                                                           constraints,
                                                           MappingQ1<dim>());
    constraints.close();

    const QGauss<dim>      quadrature_formula(fe_degree + 2);
    FEValues<dim>          fe_values(dof_handler_velocity.get_fe(),
                            quadrature_formula,
                            update_values | update_quadrature_points |
                              update_JxW_values);
    FEInterfaceValues<dim> fe_interface_values(
      dof_handler_velocity.get_fe(),
      QGauss<dim - 1>(fe_degree + 2),
      update_values | update_gradients | update_quadrature_points |
        update_hessians | update_JxW_values | update_normal_vectors);

    const unsigned int dofs_per_cell =
      dof_handler_velocity.get_fe().n_dofs_per_cell();

    const unsigned int        n_q_points = quadrature_formula.size();
    Vector<double>            cell_rhs(dofs_per_cell);
    std::vector<unsigned int> local_dof_indices(dofs_per_cell);

    auto begin = dof_handler_velocity.begin_mg(
      dof_handler.get_triangulation().n_global_levels() - 1);
    auto end = dof_handler_velocity.end_mg(
      dof_handler.get_triangulation().n_global_levels() - 1);

    const FEValuesExtractors::Vector velocities(0);

    for (auto cell = begin; cell != end; ++cell)
      if (cell->is_locally_owned_on_level())
        {
          cell_rhs = 0;
          fe_values.reinit(cell);

          std::vector<Tensor<1, dim>> load_values;
          const auto &q_points = fe_values.get_quadrature_points();
          std::transform(q_points.cbegin(),
                         q_points.cend(),
                         std::back_inserter(load_values),
                         [&](const auto &x_q) {
                           Tensor<1, dim> value;
                           for (auto c = 0U; c < dim; ++c)
                             value[c] = rhs_function.value(x_q, c);
                           return value;
                         });

          for (unsigned int q_index = 0; q_index < n_q_points; ++q_index)
            {
              for (unsigned int i = 0; i < dofs_per_cell; ++i)
                cell_rhs(i) += (fe_values[velocities].value(i, q_index) *
                                load_values[q_index] * fe_values.JxW(q_index));
            }

          cell->get_mg_dof_indices(local_dof_indices);
          constraints.distribute_local_to_global(cell_rhs,
                                                 local_dof_indices,
                                                 system_rhs_host);
        }

    for (auto cell = begin; cell != end; ++cell)
      if (cell->is_locally_owned_on_level())
        {
          for (const unsigned int face_no : cell->face_indices())
            if (cell->at_boundary(face_no))
              {
                fe_interface_values.reinit(cell, face_no);

                const unsigned int n_interface_dofs =
                  fe_interface_values.n_current_interface_dofs();
                Vector<double> cell_rhs_face(n_interface_dofs);
                cell_rhs_face = 0;

                const auto &q_points =
                  fe_interface_values.get_quadrature_points();
                const std::vector<double> &JxW =
                  fe_interface_values.get_JxW_values();
                const std::vector<Tensor<1, dim>> &normals =
                  fe_interface_values.get_normal_vectors();

                std::vector<Tensor<1, dim>> tangential_solution_values;
                std::vector<Tensor<1, dim>> solution_values;
                std::transform(q_points.cbegin(),
                               q_points.cend(),
                               std::back_inserter(solution_values),
                               [&](const auto &x_q) {
                                 Tensor<1, dim> value;
                                 for (auto c = 0U; c < dim; ++c)
                                   value[c] = exact_solution.value(x_q, c);
                                 return value;
                               });
                std::transform(solution_values.cbegin(),
                               solution_values.cend(),
                               normals.cbegin(),
                               std::back_inserter(tangential_solution_values),
                               [](const auto &u_q, const auto &normal) {
                                 return u_q - ((u_q * normal) * normal);
                               });

                const unsigned int p = fe_degree;
                const auto         h = cell->extent_in_direction(
                  GeometryInfo<dim>::unit_normal_direction[face_no]);
                const auto   one_over_h   = (0.5 / h) + (0.5 / h);
                const auto   gamma        = p == 0 ? 1 : p * (p + 1);
                const double gamma_over_h = 2.0 * gamma * one_over_h;

                for (unsigned int qpoint = 0; qpoint < q_points.size();
                     ++qpoint)
                  {
                    const auto &n = normals[qpoint];

                    for (unsigned int i = 0; i < n_interface_dofs; ++i)
                      {
                        const auto av_gradients_i_dot_n_dot_n =
                          (fe_interface_values.average_of_shape_gradients(
                             i, qpoint) *
                           n * n);
                        const auto jump_val_i_dot_n =
                          (fe_interface_values.jump_in_shape_values(i, qpoint) *
                           n);
                        cell_rhs_face(i) +=
                          (-av_gradients_i_dot_n_dot_n * // - {grad v n n }
                             (tangential_solution_values[qpoint]) //   (u_exact
                                                                  //   . n)
                           +                                      // +
                           gamma_over_h                           //  gamma/h
                             * jump_val_i_dot_n                   // [v n]
                             * (tangential_solution_values[qpoint]) // (u_exact
                                                                    // . n)
                           ) *
                          JxW[qpoint]; // dx
                      }
                  }

                auto dof_indices =
                  fe_interface_values.get_interface_dof_indices();
                constraints.distribute_local_to_global(cell_rhs_face,
                                                       dof_indices,
                                                       system_rhs_host);
              }
        }

    system_rhs_host.compress(VectorOperation::add);
    rw_vector.import(system_rhs_host, VectorOperation::insert);
    system_rhs_dev.import(rw_vector, VectorOperation::insert);

    // system_rhs_dev.print(std::cout);

    *pcout << "RHS setup time:         " << time.wall_time() << "s"
           << std::endl;
  }
  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::assemble_mg()
  {
    // Initialization of Dirichlet boundaries
    std::set<types::boundary_id> dirichlet_boundary;
    dirichlet_boundary.insert(0);
    mg_constrained_dofs.initialize(dof_handler_velocity);
    mg_constrained_dofs.make_zero_boundary_constraints(dof_handler_velocity,
                                                       dirichlet_boundary);

    // set up a mapping for the geometry representation
    MappingQ1<dim> mapping;

    unsigned int minlevel = 1;
    unsigned int maxlevel = triangulation.n_global_levels() - 1;

    mfdata_dp.resize(1, maxlevel);

    if (std::is_same_v<vcycle_number, float>)
      mfdata_sp.resize(1, maxlevel);

    Timer time;
    for (unsigned int level = minlevel; level <= maxlevel; ++level)
      {
        // IndexSet relevant_dofs;
        // DoFTools::extract_locally_relevant_level_dofs(dof_handler,
        //                                               level,
        //                                               relevant_dofs);
        // double-precision matrix-free data
        {
          // AffineConstraints<full_number> level_constraints;
          // level_constraints.reinit(relevant_dofs);
          // level_constraints.add_lines(
          //   mg_constrained_dofs.get_boundary_indices(level));
          // level_constraints.close();

          typename MatrixFreeDP::AdditionalData additional_data;
          additional_data.relaxation         = 1.;
          additional_data.use_coloring       = false;
          additional_data.patch_per_block    = CT::PATCH_PER_BLOCK_;
          additional_data.granularity_scheme = CT::GRANULARITY_;

          mfdata_dp[level] = std::make_shared<MatrixFreeDP>();
          mfdata_dp[level]->reinit(dof_handler_velocity,
                                   dof_handler_pressure,
                                   mg_constrained_dofs,
                                   level,
                                   additional_data);
        }

        // single-precision matrix-free data
        if (std::is_same_v<vcycle_number, float>)
          {
            // AffineConstraints<vcycle_number> level_constraints;
            // level_constraints.reinit(relevant_dofs);
            // level_constraints.add_lines(
            //   mg_constrained_dofs.get_boundary_indices(level));
            // level_constraints.close();

            typename MatrixFreeSP::AdditionalData additional_data;
            additional_data.relaxation         = 1.;
            additional_data.use_coloring       = false;
            additional_data.patch_per_block    = CT::PATCH_PER_BLOCK_;
            additional_data.granularity_scheme = CT::GRANULARITY_;

            mfdata_sp[level] = std::make_shared<MatrixFreeSP>();
            mfdata_sp[level]->reinit(dof_handler_velocity,
                                     dof_handler_pressure,
                                     mg_constrained_dofs,
                                     level,
                                     additional_data);
          }
      }

    *pcout << "Matrix-free setup time: " << time.wall_time() << "s"
           << std::endl;

    time.restart();

    transfer.initialize_constraints(mg_constrained_dofs);
    transfer.build(dof_handler_velocity, dof_handler_pressure);

    *pcout << "MG transfer setup time: " << time.wall_time() << "s"
           << std::endl;
  }

  template <int dim, int fe_degree>
  template <PSMF::LaplaceVariant     laplace,
            PSMF::LaplaceVariant     smooth_vmult,
            PSMF::SmootherVariant    smooth_inv,
            PSMF::LocalSolverVariant local_solver>
  void
  LaplaceProblem<dim, fe_degree>::do_solve(unsigned int k,
                                           unsigned int j,
                                           unsigned int i,
                                           unsigned int call_count)
  {
    PSMF::MultigridSolver<dim,
                          fe_degree,
                          full_number,
                          local_solver,
                          laplace,
                          smooth_vmult,
                          smooth_inv,
                          vcycle_number>
      solver(dof_handler,
             dof_handler_velocity,
             mfdata_dp,
             mfdata_sp,
             transfer,
             system_rhs_dev,
             pcout,
             1);

    *pcout << "\nMG with [" << LaplaceToString(CT::LAPLACE_TYPE_[k]) << " "
           << LaplaceToString(CT::SMOOTH_VMULT_[k]) << " "
           << SmootherToString(CT::SMOOTH_INV_[j]) << " "
           << LocalSolverToString(CT::LOCAL_SOLVER_[i]) << "]\n";

    unsigned int index =
      (k * CT::SMOOTH_INV_.size() + j) * CT::LOCAL_SOLVER_.size() + i;

    info_table[index].add_value("level", triangulation.n_global_levels());
    info_table[index].add_value("cells", triangulation.n_global_active_cells());
    info_table[index].add_value("dofs", dof_handler.n_dofs());
    info_table[index].add_value("dofs_v", dof_handler_velocity.n_dofs());
    info_table[index].add_value("dofs_p", dof_handler_pressure.n_dofs());

    std::vector<PSMF::SolverData> comp_data = solver.static_comp();
    for (auto &data : comp_data)
      {
        *pcout << data.print_comp();

        auto times = data.solver_name + "[s]";
        auto perfs = data.solver_name + "Perf[Dof/s]";

        info_table[index].add_value(times, data.timing);
        info_table[index].add_value(perfs, data.perf);

        if (call_count == 0)
          {
            info_table[index].set_scientific(times, true);
            info_table[index].set_precision(times, 3);
            info_table[index].set_scientific(perfs, true);
            info_table[index].set_precision(perfs, 3);

            info_table[index].add_column_to_supercolumn(times,
                                                        data.solver_name);
            info_table[index].add_column_to_supercolumn(perfs,
                                                        data.solver_name);
          }
      }

    *pcout << std::endl;

    std::vector<PSMF::SolverData> solver_data = solver.solve();
    for (auto &data : solver_data)
      {
        *pcout << data.print_solver();

        auto it    = data.solver_name + "it";
        auto step  = data.solver_name + "step";
        auto times = data.solver_name + "[s]";
        auto mem   = data.solver_name + "Mem Usage[MB]";

        info_table[index].add_value(it, data.n_iteration);
        info_table[index].add_value(step, data.n_step);
        info_table[index].add_value(times, data.timing);
        info_table[index].add_value(mem, data.mem_usage);

        if (call_count == 0)
          {
            info_table[index].set_scientific(times, true);
            info_table[index].set_precision(times, 3);

            info_table[index].add_column_to_supercolumn(it, data.solver_name);
            info_table[index].add_column_to_supercolumn(step, data.solver_name);
            info_table[index].add_column_to_supercolumn(times,
                                                        data.solver_name);
            info_table[index].add_column_to_supercolumn(mem, data.solver_name);
          }
      }

    if (CT::SETS_ == "error_analysis")
      {
        auto solution = solver.get_solution();

        LinearAlgebra::distributed::Vector<double, MemorySpace::Host>
                                               solution_host(solution.size());
        LinearAlgebra::ReadWriteVector<double> rw_vector(solution.size());
        rw_vector.import(solution, VectorOperation::insert);
        solution_host.import(rw_vector, VectorOperation::insert);

        solution_velocity_host.reinit(dof_handler_velocity.n_dofs());
        solution_pressure_host.reinit(dof_handler_pressure.n_dofs());

        for (unsigned int i = 0; i < solution_velocity_host.size(); ++i)
          solution_velocity_host[i] = solution_host[i];

        for (unsigned int i = 0; i < solution_pressure_host.size(); ++i)
          solution_pressure_host[i] =
            solution_host[solution_velocity_host.size() + i];

        constraints.distribute(solution_velocity_host);

        // solution_host.print(std::cout);
        double l2_error_v = 0;
        double l2_error_p = 0;
        double H1_error_v = 0;

        if (dof_handler.n_dofs() < 5e6)
          {
            const auto [l2_error, l2_error_, H1_error] = compute_error();

            l2_error_v = l2_error;
            l2_error_p = l2_error_;
            H1_error_v = H1_error;
          }

        *pcout << "L2 error velocity: " << l2_error_v << std::endl
               << "L2 error pressure: " << l2_error_p << std::endl
               << "H1 error velocity: " << H1_error_v << std::endl
               << std::endl;

        // ghost_solution_host.print(std::cout);

        info_table[index].add_value("l2_error_v", l2_error_v);
        info_table[index].set_scientific("l2_error_v", true);
        info_table[index].set_precision("l2_error_v", 3);

        info_table[index].evaluate_convergence_rates(
          "l2_error_v", "dofs", ConvergenceTable::reduction_rate_log2, dim);

        info_table[index].add_value("l2_error_p", l2_error_p);
        info_table[index].set_scientific("l2_error_p", true);
        info_table[index].set_precision("l2_error_p", 3);

        info_table[index].evaluate_convergence_rates(
          "l2_error_p", "dofs", ConvergenceTable::reduction_rate_log2, dim);

        info_table[index].add_value("H1_error_v", H1_error_v);
        info_table[index].set_scientific("H1_error_v", true);
        info_table[index].set_precision("H1_error_v", 3);

        info_table[index].evaluate_convergence_rates(
          "H1_error_v", "dofs", ConvergenceTable::reduction_rate_log2, dim);
      }
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::solve_mg(unsigned int n_mg_cycles)
  {
    // static unsigned int call_count = 0;

    Tester<CT::LAPLACE_TYPE_.size() - 1,
           CT::SMOOTH_INV_.size() - 1,
           CT::LOCAL_SOLVER_.size() - 1>::run(*this);

    // do_solve<CT::LAPLACE_TYPE_[0],
    //          CT::SMOOTH_VMULT_[0],
    //          CT::SMOOTH_INV_[0],
    //          CT::LOCAL_SOLVER_[0]>(0, 0, 0, call_count);

    call_count++;
  }

  template <int dim, int fe_degree>
  std::tuple<double, double, double>
  LaplaceProblem<dim, fe_degree>::compute_error()
  {
    const double mean_pressure =
      VectorTools::compute_mean_value(dof_handler_pressure,
                                      QGauss<dim>(fe_degree + 2),
                                      solution_pressure_host,
                                      0);
    solution_pressure_host.add(-mean_pressure);
    *pcout << "\nNote: The mean value was adjusted by " << -mean_pressure
           << std::endl;

    Vector<double> cellwise_norm(triangulation.n_active_cells());
    VectorTools::integrate_difference(dof_handler_velocity,
                                      solution_velocity_host,
                                      SolutionVelocity<dim>(),
                                      cellwise_norm,
                                      QGauss<dim>(fe->degree + 2),
                                      VectorTools::L2_norm);
    const double global_norm_v =
      VectorTools::compute_global_error(triangulation,
                                        cellwise_norm,
                                        VectorTools::L2_norm);

    Vector<double> cellwise_norm_p(triangulation.n_active_cells());
    VectorTools::integrate_difference(dof_handler_pressure,
                                      solution_pressure_host,
                                      SolutionPressure<dim>(),
                                      cellwise_norm_p,
                                      QGauss<dim>(fe->degree + 2),
                                      VectorTools::L2_norm);
    const double global_norm_p =
      VectorTools::compute_global_error(triangulation,
                                        cellwise_norm_p,
                                        VectorTools::L2_norm);

    Vector<double> cellwise_h1norm(triangulation.n_active_cells());
    VectorTools::integrate_difference(dof_handler_velocity,
                                      solution_velocity_host,
                                      SolutionVelocity<dim>(),
                                      cellwise_h1norm,
                                      QGauss<dim>(fe->degree + 2),
                                      VectorTools::H1_seminorm);
    const double global_h1norm =
      VectorTools::compute_global_error(triangulation,
                                        cellwise_h1norm,
                                        VectorTools::H1_seminorm);

    return std::make_tuple(global_norm_v, global_norm_p, global_h1norm);
  }

  template <int dim, int fe_degree>
  void
  LaplaceProblem<dim, fe_degree>::run(const unsigned int n_cycles)
  {
    *pcout << Util::generic_info_to_fstring() << std::endl;

    for (unsigned int cycle = 0; cycle < n_cycles; ++cycle)
      {
        *pcout << "Cycle " << cycle << std::endl;

        unsigned int n_levels = triangulation.n_global_levels();

        long long unsigned int n_dofs =
          (dim + 1) * std::pow(std::pow(2, n_levels) * (fe_degree + 1), dim);

        if (n_dofs > CT::MAX_SIZES_)
          {
            *pcout << "Max size reached, terminating." << std::endl;
            *pcout << std::endl;

            break;
          }

        if (cycle == 0)
          {
            GridGenerator::hyper_cube(triangulation, 0., 1.);
            triangulation.refine_global(2);
          }
        else
          triangulation.refine_global(1);

        setup_system();
        assemble_rhs();
        assemble_mg();

        solve_mg(1);
        *pcout << std::endl;
      }

    {
      for (unsigned int k = 0; k < CT::LAPLACE_TYPE_.size(); ++k)
        for (unsigned int j = 0; j < CT::SMOOTH_INV_.size(); ++j)
          for (unsigned int i = 0; i < CT::LOCAL_SOLVER_.size(); ++i)
            {
              unsigned int index =
                (k * CT::SMOOTH_INV_.size() + j) * CT::LOCAL_SOLVER_.size() + i;

              std::ostringstream oss;

              oss << "\n[" << LaplaceToString(CT::LAPLACE_TYPE_[k]) << " "
                  << LaplaceToString(CT::SMOOTH_VMULT_[k]) << " "
                  << SmootherToString(CT::SMOOTH_INV_[j]) << " "
                  << LocalSolverToString(CT::LOCAL_SOLVER_[i]) << "]\n";
              info_table[index].write_text(oss);

              *pcout << oss.str() << std::endl;
            }
    }
  }
} // namespace Step64
int
main(int argc, char *argv[])
{
  try
    {
      using namespace Step64;

      {
        int device_id = findCudaDevice(argc, (const char **)argv);
        AssertCuda(hipSetDevice(device_id));
      }

      {
        LaplaceProblem<CT::DIMENSION_, CT::FE_DEGREE_> Laplace_problem;
        Laplace_problem.run(20);
      }
    }
  catch (std::exception &exc)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Exception on processing: " << std::endl
                << exc.what() << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  catch (...)
    {
      std::cerr << std::endl
                << std::endl
                << "----------------------------------------------------"
                << std::endl;
      std::cerr << "Unknown exception!" << std::endl
                << "Aborting!" << std::endl
                << "----------------------------------------------------"
                << std::endl;
      return 1;
    }
  return 0;
}
