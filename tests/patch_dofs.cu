#include "hip/hip_runtime.h"
/**
 * Created by Cu Cui on 2023/4/17.
 */

// Testing local patch dofs numbering

#include <deal.II/base/polynomials_raviart_thomas.h>
#include <deal.II/base/quadrature_lib.h>

#include <deal.II/dofs/dof_renumbering.h>
#include <deal.II/dofs/dof_tools.h>

#include <deal.II/fe/fe_dgq.h>
#include <deal.II/fe/fe_raviart_thomas_new.h>
#include <deal.II/fe/fe_tools.h>

#include <deal.II/grid/grid_generator.h>
#include <deal.II/grid/tria.h>

#include <deal.II/matrix_free/shape_info.h>

#include <iostream>

#include "TPSS/tensor_product_matrix.h"
#include "renumber.h"
#include "utilities.cuh"

using namespace dealii;

template <int dim>
std::vector<unsigned int>
get_lexicographic_numbering(const unsigned int normal_degree,
                            const unsigned int tangential_degree)
{
  const unsigned int n_dofs_face =
    Utilities::pow(tangential_degree + 1, dim - 1);
  std::vector<unsigned int> lexicographic_numbering;
  // component 1
  for (unsigned int j = 0; j < n_dofs_face; ++j)
    {
      lexicographic_numbering.push_back(j);
      if (normal_degree > 1)
        for (unsigned int i = n_dofs_face * 2 * dim;
             i < n_dofs_face * 2 * dim + normal_degree - 1;
             ++i)
          lexicographic_numbering.push_back(i + j * (normal_degree - 1));
      lexicographic_numbering.push_back(n_dofs_face + j);
    }

  std::cout << lexicographic_numbering.size() << std::endl;

  // component 2
  unsigned int layers = (dim == 3) ? tangential_degree + 1 : 1;
  for (unsigned int k = 0; k < layers; ++k)
    for (unsigned int j = 0; j < tangential_degree + 1; ++j)
      {
        unsigned int s = j + n_dofs_face * 2;

        unsigned int k_add = k * (tangential_degree + 1);

        lexicographic_numbering.push_back(s + k_add);

        if (normal_degree > 1)
          for (unsigned int i = n_dofs_face * (2 * dim + (normal_degree - 1));
               i < n_dofs_face * (2 * dim + (normal_degree - 1)) +
                     (normal_degree - 1) * (tangential_degree + 1);
               i += tangential_degree + 1)
            {
              lexicographic_numbering.push_back(i + j +
                                                k_add * tangential_degree);
            }
        unsigned int e = j + n_dofs_face * 3;
        lexicographic_numbering.push_back(e + k_add);
      }

  std::cout << lexicographic_numbering.size() << std::endl;
  // unsigned int layers = (dim == 3) ? tangential_degree + 1 : 1;
  // for (unsigned int k = 0; k < layers; ++k)
  //   {
  //     unsigned int k_add = k * (tangential_degree + 1);
  //     for (unsigned int j = n_dofs_face * 2;
  //          j < n_dofs_face * 2 + tangential_degree + 1;
  //          ++j)
  //       lexicographic_numbering.push_back(j + k_add);

  //     if (normal_degree > 1)
  //       for (unsigned int i = n_dofs_face * (2 * dim + (normal_degree - 1));
  //            i < n_dofs_face * (2 * dim + (normal_degree - 1)) +
  //                  (normal_degree - 1) * (tangential_degree + 1);
  //            ++i)
  //         {
  //           lexicographic_numbering.push_back(i + k_add * tangential_degree);
  //         }
  //     for (unsigned int j = n_dofs_face * 3;
  //          j < n_dofs_face * 3 + tangential_degree + 1;
  //          ++j)
  //       lexicographic_numbering.push_back(j + k_add);
  //   }

  // component 3
  if (dim == 3)
    {
      for (unsigned int k = 0; k < layers; ++k)
        for (unsigned int j = 0; j < tangential_degree + 1; ++j)
          {
            unsigned int k_add = k * (tangential_degree + 1);

            unsigned int s = j + 4 * n_dofs_face;
            lexicographic_numbering.push_back(s + k_add);

            if (normal_degree > 1)
              {
                for (unsigned int i =
                       6 * n_dofs_face + n_dofs_face * 2 * (normal_degree - 1);
                     i < 6 * n_dofs_face +
                           n_dofs_face * 2 * (normal_degree - 1) +
                           (normal_degree - 1) * (tangential_degree + 1);
                     i += tangential_degree + 1)
                  lexicographic_numbering.push_back(i + j +
                                                    k_add * tangential_degree);
              }

            unsigned int e = j + 5 * n_dofs_face;
            lexicographic_numbering.push_back(e + k_add);
          }

      std::cout << lexicographic_numbering.size() << std::endl;

      // for (unsigned int i = 4 * n_dofs_face; i < 5 * n_dofs_face; ++i)
      //   lexicographic_numbering.push_back(i);
      // if (normal_degree > 1)
      //   for (unsigned int i =
      //          6 * n_dofs_face + n_dofs_face * 2 * (normal_degree - 1);
      //        i < 6 * n_dofs_face + n_dofs_face * 3 * (normal_degree - 1);
      //        ++i)
      //     lexicographic_numbering.push_back(i);
      // for (unsigned int i = 5 * n_dofs_face; i < 6 * n_dofs_face; ++i)
      //   lexicographic_numbering.push_back(i);
    }

  return lexicographic_numbering;
}

template <int dim, int degree>
std::vector<types::global_dof_index>
patch_dofs_numbering(std::array<std::vector<unsigned int>, 1 << dim> &cell_dofs)
{
  std::array<std::vector<unsigned int>, dim> cell_number;

  if (dim == 2)
    {
      cell_number[0] = {{0, 1, 2, 3}};
      cell_number[1] = {{0, 2, 1, 3}};
    }
  else if (dim == 3)
    {
      cell_number[0] = {{0, 1, 2, 3, 4, 5, 6, 7}};
      cell_number[1] = {{0, 2, 1, 3, 4, 6, 5, 7}};
      cell_number[2] = {{0, 4, 1, 5, 2, 6, 3, 7}};
    }

  std::vector<types::global_dof_index> local_dof_indices;

  const unsigned int layer = dim == 2 ? 1 : degree + 1;
  const unsigned int n_z   = dim == 2 ? 1 : 2;

  for (auto d = 0U; d < dim; ++d)
    for (auto z = 0U; z < n_z; ++z)
      for (auto l = 0U; l < layer; ++l)
        for (auto row = 0U; row < 2; ++row)
          for (auto i = 0U; i < degree + 1; ++i)
            for (auto col = 0U; col < 2; ++col)
              for (auto k = 0U; k < degree + 2; ++k)
                {
                  if (k == 0 && col == 1)
                    continue;

                  const unsigned int cell =
                    cell_number[d][z * 4 + row * 2 + col];

                  local_dof_indices.push_back(
                    cell_dofs[cell][d * Util::pow(degree + 1, dim - 1) *
                                      (degree + 2) +
                                    l * (degree + 1) * (degree + 2) +
                                    i * (degree + 2) + k]);
                }

  return local_dof_indices;
}

template <int dim, int degree>
std::vector<types::global_dof_index>
patch_dofs_numbering_interior(
  std::vector<types::global_dof_index> &patch_numbering)
{
  const unsigned int n_comp_dofs = patch_numbering.size() / dim;

  constexpr unsigned int n_dofs_normal = 2 * degree + 3;
  constexpr unsigned int n_dofs_tang   = 2 * degree + 2;

  constexpr unsigned int n_z = dim == 2 ? 1 : 2 * degree;

  std::vector<types::global_dof_index> local_dof_indices;

  for (auto d = 0U; d < dim; ++d)
    for (auto i = 0U; i < n_z; ++i)
      for (auto j = 0U; j < 2 * degree; ++j)
        for (auto k = 0U; k < 2 * degree + 1; ++k)
          {
            local_dof_indices.push_back(
              patch_numbering[d * n_comp_dofs +
                              (i + dim - 2) * n_dofs_normal * n_dofs_tang +
                              (j + 1) * n_dofs_normal + k + 1]);
          }

  return local_dof_indices;
}

std::vector<types::global_dof_index>
reverse_numbering(std::vector<types::global_dof_index> &l_numbering)
{
  auto sortedVector = l_numbering;
  std::sort(sortedVector.begin(), sortedVector.end());

  std::vector<types::global_dof_index> local_dof_indices(l_numbering.size());
  for (auto i = 0U; i < l_numbering.size(); ++i)
    {
      auto it =
        std::find(l_numbering.begin(), l_numbering.end(), sortedVector[i]);

      local_dof_indices[i] = std::distance(l_numbering.begin(), it);
    }
  return local_dof_indices;
}

template <int dim, int degree, typename Number = double>
void
test()
{
  FE_RaviartThomas_new<dim> fe_v(degree);
  FE_DGQLegendre<dim>       fe_p(degree);
  QGauss<1>                 quadrature(degree + 2);

  auto numbering = get_lexicographic_numbering<dim>(degree + 1, degree);

  for (auto ind : numbering)
    std::cout << ind << ", ";
  std::cout << std::endl;

  Triangulation<dim> triangulation(
    Triangulation<dim>::limit_level_difference_at_vertices);
  GridGenerator::hyper_cube(triangulation, 0., 1.);
  triangulation.refine_global(1);

  DoFHandler<dim> dof_handler_v(triangulation);
  dof_handler_v.distribute_dofs(fe_v);

  DoFHandler<dim> dof_handler_p(triangulation);
  dof_handler_p.distribute_dofs(fe_p);

  {
    std::cout << "RT\n";
    const unsigned int                   dofs_per_cell = fe_v.n_dofs_per_cell();
    std::vector<types::global_dof_index> local_dof_indices(dofs_per_cell);

    std::array<std::vector<unsigned int>, 1 << dim> cell_dofs;

    std::set<unsigned int> h_numbering;

    unsigned int c = 0;

    std::cout << std::endl;
    for (const auto &cell : dof_handler_v.active_cell_iterators())
      {
        cell->get_dof_indices(local_dof_indices);

        for (auto ind : local_dof_indices)
          {
            h_numbering.insert(ind);
            std::cout << ind << ", ";
          }
        std::cout << std::endl;

        for (auto i = 0U; i < dofs_per_cell; ++i)
          std::cout << local_dof_indices[numbering[i]] << ", ";
        std::cout << std::endl;

        std::cout << std::endl;

        cell_dofs[c].resize(dofs_per_cell);
        for (auto i = 0U; i < dofs_per_cell; ++i)
          cell_dofs[c][i] = local_dof_indices[numbering[i]];

        c++;
      }

    std::cout << std::endl;

    std::vector<unsigned int> patch_h_numbering(h_numbering.begin(),
                                                h_numbering.end());

    for (auto i : patch_h_numbering)
      std::cout << i << " ";
    std::cout << std::endl;

    auto patch_numbering = patch_dofs_numbering<dim, degree>(cell_dofs);

    for (auto i : patch_numbering)
      std::cout << i << " ";
    std::cout << std::endl;

    auto patch_numbering_interior =
      patch_dofs_numbering_interior<dim, degree>(patch_numbering);

    for (auto i : patch_numbering_interior)
      std::cout << i << " ";
    std::cout << std::endl << std::endl;

    auto l_to_h = reverse_numbering(patch_numbering);
    for (auto i : l_to_h)
      std::cout << i << " ";
    std::cout << std::endl;

    auto l_to_h_int = reverse_numbering(patch_numbering_interior);
    for (auto i : l_to_h_int)
      std::cout << i << " ";
    std::cout << std::endl;
  }

  std::cout << std::endl;

  {
    std::cout << "DG\n";
    const unsigned int                   dofs_per_cell = fe_p.n_dofs_per_cell();
    std::vector<types::global_dof_index> local_dof_indices(dofs_per_cell);

    std::array<std::vector<unsigned int>, 1 << dim> cell_dofs;

    std::set<unsigned int> h_numbering;

    unsigned int c = 0;

    std::cout << std::endl;
    for (const auto &cell : dof_handler_p.active_cell_iterators())
      {
        cell->get_dof_indices(local_dof_indices);

        for (auto ind : local_dof_indices)
          {
            h_numbering.insert(ind);
            std::cout << ind << ", ";
          }

        std::cout << std::endl;

        cell_dofs[c].resize(dofs_per_cell);
        for (auto i = 0U; i < dofs_per_cell; ++i)
          cell_dofs[c][i] = local_dof_indices[i];

        c++;
      }

    std::cout << std::endl;

    auto patch_dofs_numbering_normal = [&]() {
      std::vector<types::global_dof_index> local_dof_indices;

      const unsigned int layer = dim == 2 ? 1 : degree + 1;
      const unsigned int n_z   = dim == 2 ? 1 : 2;

      for (auto z = 0U; z < n_z; ++z)
        for (auto l = 0U; l < layer; ++l)
          for (auto row = 0U; row < 2; ++row)
            for (auto i = 0U; i < degree + 1; ++i)
              for (auto col = 0U; col < 2; ++col)
                for (auto k = 0U; k < degree + 1; ++k)
                  {
                    const unsigned int cell = z * 4 + row * 2 + col;

                    local_dof_indices.push_back(
                      cell_dofs[cell][l * (degree + 1) * (degree + 1) +
                                      i * (degree + 1) + k]);
                  }

      return local_dof_indices;
    };

    auto patch_dofs_numbering_tang = [&]() {
      std::vector<types::global_dof_index> local_dof_indices;

      const unsigned int layer = dim == 2 ? 1 : degree + 1;
      const unsigned int n_z   = dim == 2 ? 1 : 2;

      for (auto z = 0U; z < n_z; ++z)
        for (auto l = 0U; l < layer; ++l)
          for (auto row = 0U; row < 2; ++row)
            for (auto i = 0U; i < degree + 1; ++i)
              for (auto col = 0U; col < 2; ++col)
                for (auto k = 0U; k < degree + 1; ++k)
                  {
                    const unsigned int cell = z * 4 + row + col * 2;

                    local_dof_indices.push_back(
                      cell_dofs[cell][l * (degree + 1) * (degree + 1) + i +
                                      k * (degree + 1)]);
                  }

      return local_dof_indices;
    };

    std::vector<unsigned int> patch_h_numbering(h_numbering.begin(),
                                                h_numbering.end());

    for (auto i : patch_h_numbering)
      std::cout << i << " ";
    std::cout << std::endl;

    auto patch_numbering = patch_dofs_numbering_normal();

    for (auto i : patch_numbering)
      std::cout << i << " ";
    std::cout << std::endl;

    auto patch_numbering_tang = patch_dofs_numbering_tang();

    for (auto i : patch_numbering_tang)
      std::cout << i << " ";
    std::cout << std::endl;

    auto patch_dofs_numbering_interior = [&](auto numbering) {
      std::vector<types::global_dof_index> local_dof_indices;

      constexpr unsigned int n_z = dim == 2 ? 1 : 2 * degree;

      for (auto i = 0U; i < n_z; ++i)
        for (auto j = 0U; j < 2 * degree; ++j)
          for (auto k = 0U; k < 2 * degree; ++k)
            {
              local_dof_indices.push_back(
                numbering[(i + dim - 2) * (2 * degree + 2) * (2 * degree + 2) +
                          (j + 1) * (2 * degree + 2) + k + 1]);
            }

      return local_dof_indices;
    };

    auto patch_numbering_interior =
      patch_dofs_numbering_interior(patch_numbering);

    for (auto i : patch_numbering_interior)
      std::cout << i << " ";
    std::cout << std::endl;

    auto patch_numbering_interior_tang =
      patch_dofs_numbering_interior(patch_numbering_tang);

    for (auto i : patch_numbering_interior_tang)
      std::cout << i << " ";
    std::cout << std::endl << std::endl;

    auto l_to_h = reverse_numbering(patch_numbering);
    for (auto i : l_to_h)
      std::cout << i << " ";
    std::cout << std::endl;

    auto l_to_h_t = reverse_numbering(patch_numbering_tang);
    for (auto i : l_to_h_t)
      std::cout << i << " ";
    std::cout << std::endl;

    auto l_to_h_int = reverse_numbering(patch_numbering_interior);
    for (auto i : l_to_h_int)
      std::cout << i << " ";
    std::cout << std::endl;

    auto l_to_h_int_t = reverse_numbering(patch_numbering_interior_tang);
    for (auto i : l_to_h_int_t)
      std::cout << i << " ";
    std::cout << std::endl;
  }
}

template <int dim, int degree>
void
run()
{
  auto print_vec = [](auto &vec) {
    for (auto i : vec)
      std::cout << i << " ";
    std::cout << "\n";
  };

  PSMF::DoFMapping<dim, degree> dm;

  // {
  //   auto l = dm.get_h_to_l_rt();
  //   print_vec(l);

  //   auto l1 = dm.get_h_to_l_rt_interior();
  //   print_vec(l1);

  //   auto l2 = dm.get_l_to_h_rt();
  //   print_vec(l2);
  // }

  // auto l3 = dm.get_l_to_h_rt_interior();
  // print_vec(l3);

  // {
  //   auto l = dm.get_h_to_l_dg_normal();
  //   print_vec(l);

  //   auto l1 = dm.get_h_to_l_dg_tangent();
  //   print_vec(l1);

  auto lz = dm.get_h_to_l_dg_z();
  print_vec(lz);

  //   auto l2 = dm.get_l_to_h_dg_tangent();
  //   print_vec(l2);

  //   auto l3 = dm.get_l_to_h_dg_normal();
  //   print_vec(l3);

  auto l3z = dm.get_l_to_h_dg_z();
  print_vec(l3z);
  // }
}

template <int dim, int degree>
void
print_dofs()
{
  FESystem<dim> fe(FE_RaviartThomas_new<dim>(degree),
                   1,
                   FE_DGQLegendre<dim>(degree),
                   1);

  std::cout << fe.get_name() << "\n";

  Triangulation<dim> triangulation(
    Triangulation<dim>::limit_level_difference_at_vertices);
  GridGenerator::hyper_cube(triangulation, 0., 1.);
  triangulation.refine_global(1);

  DoFHandler<dim> dof_handler(triangulation);
  dof_handler.distribute_dofs(fe);

  DoFRenumbering::component_wise(dof_handler);

  const unsigned int                   dofs_per_cell = fe.n_dofs_per_cell();
  std::vector<types::global_dof_index> local_dof_indices(dofs_per_cell);

  for (const auto &cell : dof_handler.active_cell_iterators())
    {
      cell->get_dof_indices(local_dof_indices);

      for (auto ind : local_dof_indices)
        {
          std::cout << ind << ", ";
        }

      std::cout << std::endl;
    }

  std::cout << dof_handler.get_fe().get_sub_fe(0, dim).n_dofs_per_cell() << " "
            << dof_handler.get_fe().get_sub_fe(dim, 1).n_dofs_per_cell()
            << "\n";

  auto print_vec = [](auto &vec) {
    for (auto i : vec)
      std::cout << i << " ";
    std::cout << "\n";
  };

  PSMF::DoFMapping<dim> dm(degree);

  auto first_dof  = dm.get_first_dofs();
  auto base_dof   = dm.get_base_dof_cell();
  auto dof_offset = dm.get_dof_offset_cell();

  auto base_dofdg   = dm.get_base_dof();
  auto dof_offsetdg = dm.get_dof_offset();


  print_vec(first_dof);
  print_vec(base_dof);
  print_vec(dof_offset);
  print_vec(base_dofdg);
  print_vec(dof_offsetdg);
}

template <int dim, int degree>
void
compute_ind()
{
  constexpr int face_dofs = Util::pow(degree + 1, dim - 1);
  constexpr int quad_dofs = dim * Util::pow(degree + 1, dim - 1) * degree;


  constexpr std::array<int, 4> cell_faces     = {{4, 3, 3, 2}};
  constexpr std::array<int, 4> cell_face_dofs = {{cell_faces[0] * face_dofs,
                                                  cell_faces[1] * face_dofs,
                                                  cell_faces[2] * face_dofs,
                                                  cell_faces[3] * face_dofs}};
  constexpr std::array<int, 4> cell_dofs      = {{cell_face_dofs[0] + quad_dofs,
                                                  cell_face_dofs[1] + quad_dofs,
                                                  cell_face_dofs[2] + quad_dofs,
                                                  cell_face_dofs[3] + quad_dofs}};

  std::cout << face_dofs << " " << quad_dofs << std::endl;

  std::vector<int> input_dofs;

  int start = 0;
  for (int c = 0; c < 4; ++c)
    {
      for (int f = 0; f < cell_faces[c]; ++f)
        {
          input_dofs.push_back(start);
          start += face_dofs;
        }
      input_dofs.push_back(start);
      start += quad_dofs;
    }

  for (auto i : input_dofs)
    std::cout << i << " ";
  std::cout << std::endl;

  int base   = -1;
  int offset = -1;

  for (int tid = 0; tid < 84; ++tid)
    {
      for (int c = 0; c < 4; ++c)
        {
          int patch_dof = 0;
          for (int subc = 0; subc < c + 1; ++subc)
            patch_dof += cell_dofs[subc];

          if (tid < patch_dof) // cell
            {
              int local_tid = tid - patch_dof + cell_dofs[c];

              if (local_tid >= cell_face_dofs[c]) // quad dof
                {
                  int shift = -1;
                  for (int subc = 0; subc <= c; ++subc)
                    shift += (1 + cell_faces[subc]);
                  base   = input_dofs[shift];
                  offset = local_tid - cell_face_dofs[c];

                  goto exitLoop;
                }

              for (int f = 0; f < cell_faces[c]; ++f)
                if (local_tid < (f + 1) * face_dofs) // face dof
                  {
                    int shift = 0;
                    for (int subc = 0; subc < c; ++subc)
                      shift += (1 + cell_faces[subc]);

                    base   = input_dofs[shift + f];
                    offset = local_tid - f * face_dofs;
                    goto exitLoop;
                  }
            }
        }
    exitLoop:
      std::cout << tid << ": " << base << " " << offset << " : "
                << base + offset << std::endl;
    }
}

int
main()
{
  // test<2, 1>();
  // test<2, 2>();
  // test<3, 2>();

  // test<2, 5>();

  // run<2, 2>();
  // run<3, 2>();

  print_dofs<3, 2>();

  // compute_ind<2, 2>();
}